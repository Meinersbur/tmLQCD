#include "hip/hip_runtime.h"
/***********************************************************************
 * Copyright (C) 2010 Florian Burger 
 *
 * This file is part of tmLQCD.
 *
 * tmLQCD is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * tmLQCD is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with tmLQCD.  If not, see <http://www.gnu.org/licenses/>.
 ***********************************************************************/

/***********************************************************************
 * This provides a NVIDIA CUDA implementation of a mixed-precision solver 
************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#include "global.h"
#include "cudaglobal.h"
#include "mixed_solve.h"
#include <math.h>


extern "C" {
#include "../tm_operators.h"
#include "../linalg_eo.h"
#include "../start.h"
#include "../complex.h"
#include "../read_input.h"
#include "../geometry_eo.h"
#include "../boundary.h"
}

#define ACCUM_N 2048
#define DOTPROD_DIM 128




int g_numofgpu;


dev_su3_2v * dev_gf;
dev_su3_2v * h2d_gf;



dev_spinor* dev_spin1;
dev_spinor* dev_spin2;
dev_spinor* dev_spin3;
dev_spinor* dev_spin4;
dev_spinor* dev_spin5;
dev_spinor* dev_spinin;
dev_spinor* dev_spinout;
dev_spinor * h2d_spin;

//additional spinors for even-odd
dev_spinor* dev_spin_eo1;
dev_spinor* dev_spin_eo2;

int * nn;
int * nn_eo;
int * nn_oe;
int * eoidx_even;
int * eoidx_odd;

int * dev_nn;
int * dev_nn_eo;
int * dev_nn_oe;

int * dev_eoidx_even;
int * dev_eoidx_odd;


size_t output_size;
int* dev_grid;
float * dev_output;



REAL hostr;
REAL hostkappa;
REAL hostm;
REAL hostmu;


__device__  REAL m;
__device__  REAL mu;
__device__  REAL r=1.0; // this is implicitly assumed to be 1.0 in the host code!!!
__device__  REAL kappa;
__device__ REAL twokappamu;

__device__ dev_complex dev_k0;
__device__ dev_complex dev_k1;
__device__ dev_complex dev_k2;
__device__ dev_complex dev_k3;

__device__ dev_complex dev_mk0;
__device__ dev_complex dev_mk1;
__device__ dev_complex dev_mk2;
__device__ dev_complex dev_mk3;


__device__  int  dev_LX,dev_LY,dev_LZ,dev_T,dev_VOLUME;


 /* texture for gauge field */
 texture<float4,1, hipReadModeElementType> gf_tex;
 const textureReference* gf_texRefPtr = NULL;
 hipChannelFormatDesc gf_channelDesc;
 
 /* texture for spinor field */
 texture<float4,1, hipReadModeElementType> spin_tex;
 const textureReference* spin_texRefPtr = NULL;
 hipChannelFormatDesc spin_channelDesc;

 /* texture for spinor field 2*/
 texture<float4,1, hipReadModeElementType> spin_tex2;
 const textureReference* spin_texRefPtr2 = NULL;
 hipChannelFormatDesc spin_channelDesc2;

__device__ inline dev_complex dev_cconj (dev_complex c){ /*konjugiert komplexe Zahl*/
 dev_complex erg;
 erg.re = c.re;
 erg.im = -1.0*c.im;
return erg;
}

__device__ inline void dev_ccopy(dev_complex* von, dev_complex* nach){/*kopiert complex von nach complex nach*/
  nach->re = von->re;
  nach->im = von->im;
}

__device__ inline REAL dev_cabssquare (dev_complex c){ /*gibt abs^2 einer komplexen Zahl zurück*/
 return c.re*c.re + c.im*c.im;
}

__device__ inline REAL dev_cabsolute (dev_complex c){/*gibt Betrag einer kompl. zahl zurück*/
 return sqrt(c.re*c.re + c.im*c.im);
}


__device__ inline  dev_complex dev_crealmult(dev_complex c1, REAL real){ /*multipliziert c1 mit reeller zahl re*/
  dev_complex erg;
  erg.re = real*c1.re;
  erg.im = real*c1.im;
return erg;
}

__device__ inline dev_complex dev_cmult (dev_complex c1, dev_complex c2){ /*multiplizier zwei komplexe Zahlen*/
  dev_complex erg;
  erg.re = c1.re * c2.re - c1.im * c2.im;
  erg.im = c1.re * c2.im + c1.im * c2.re;
return erg;
}

__device__ inline dev_complex dev_cadd (dev_complex c1, dev_complex c2){ /*addiert zwei komplexe Zahlen */
  dev_complex erg;
  erg.re = c1.re + c2.re;
  erg.im = c1.im + c2.im;
return erg;
}


__device__ inline dev_complex dev_cdiv(dev_complex c1, dev_complex c2) { /* dividiert c1 durch c2 */
  dev_complex erg;
  REAL oneovernenner = 1.0/(c2.re*c2.re + c2.im*c2.im);
  erg.re = oneovernenner*(c1.re*c2.re + c1.im*c2.im);
  erg.im = oneovernenner*(c1.im*c2.re - c1.re*c2.im);
return erg;
}


__device__ inline dev_complex dev_csub(dev_complex c1, dev_complex c2){
   dev_complex erg;
   erg.re = c1.re - c2.re;
   erg.im = c1.im - c2.im;
return erg;
}


__device__ inline dev_complex dev_initcomplex(REAL re, REAL im){/* gibt komplexe Zahl mit Realt re und Imt im zurück*/
    dev_complex erg;
    erg.re = re;
    erg.im = im;
return (erg);
}





__device__ inline void dev_copy_spinor(dev_spinor *i1, dev_spinor *i2){
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    (*(i2+i)).x = (*(i1+i)).x;
    (*(i2+i)).y = (*(i1+i)).y;
    (*(i2+i)).z = (*(i1+i)).z;
    (*(i2+i)).w = (*(i1+i)).w;
  }
}

__device__ inline void dev_zero_spinor(dev_spinor *sin){
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    (*(sin+i)).x = 0.0;
    (*(sin+i)).y = 0.0;
    (*(sin+i)).z = 0.0;
    (*(sin+i)).w = 0.0;
  }
}






//out = in + lambda in2
__device__ inline void dev_skalarmult_add_assign_spinor(dev_spinor *in, REAL lambda,dev_spinor * in2, dev_spinor * out){
  int i; 
  #pragma unroll 6
for(i=0;i<6;i++){ //color + spin
    (*(out+i)).x = (*(in+i)).x + lambda* (*(in2+i)).x;
    (*(out+i)).y = (*(in+i)).y + lambda* (*(in2+i)).y;
    (*(out+i)).z = (*(in+i)).z + lambda* (*(in2+i)).z;
    (*(out+i)).w = (*(in+i)).w + lambda* (*(in2+i)).w;
  }
}




//out = in + lambda in2
__device__ inline void dev_complexmult_add_assign_spinor(dev_spinor * in, dev_complex lambda,dev_spinor * in2, dev_spinor * out){
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    (*(out+i)).x = (*(in+i)).x + ((*(in2+i)).x*lambda.re - (*(in2+i)).y*lambda.im);
    (*(out+i)).y = (*(in+i)).y + ((*(in2+i)).x*lambda.im + (*(in2+i)).y*lambda.re);
    (*(out+i)).z = (*(in+i)).z + ((*(in2+i)).z*lambda.re - (*(in2+i)).w*lambda.im);
    (*(out+i)).w = (*(in+i)).w + ((*(in2+i)).z*lambda.im + (*(in2+i)).w*lambda.re);
  }
}




//out = in + (lambda)* in2
__device__ inline void dev_complexcgmult_add_assign_spinor(dev_spinor * in, dev_complex lambda,dev_spinor * in2, dev_spinor * out){
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    (*(out+i)).x = (*(in+i)).x + ((*(in2+i)).x*lambda.re + (*(in2+i)).y*lambda.im);
    (*(out+i)).y = (*(in+i)).y + (-(*(in2+i)).x*lambda.im + (*(in2+i)).y*lambda.re);
    (*(out+i)).z = (*(in+i)).z + ((*(in2+i)).z*lambda.re + (*(in2+i)).w*lambda.im);
    (*(out+i)).w = (*(in+i)).w + (-(*(in2+i)).z*lambda.im + (*(in2+i)).w*lambda.re);
  }
}



__device__ void inline dev_skalarmult_spinor(dev_spinor * in, dev_complex lambda, dev_spinor * out){
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    //out[i] = dev_cmult(in[i],lambda);
    
    (*(out+i)).x = (*(in+i)).x*lambda.re - (*(in+i)).y*lambda.im;
    (*(out+i)).y = (*(in+i)).y*lambda.re + (*(in+i)).x*lambda.im;
    
    (*(out+i)).z = (*(in+i)).z*lambda.re - (*(in+i)).w*lambda.im;
    (*(out+i)).w = (*(in+i)).w*lambda.re + (*(in+i)).z*lambda.im;
  }
}



__device__ void inline dev_realmult_spinor(dev_spinor * in, REAL lambda){
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    //in[i] = in[i]*lambda;
    (*(in+i)).x = (*(in+i)).x*lambda;
    (*(in+i)).y = (*(in+i)).y*lambda;
    
    (*(in+i)).z = (*(in+i)).z*lambda;
    (*(in+i)).w = (*(in+i)).w*lambda;
  }
}





__device__ inline void dev_add_spinor_assign(dev_spinor * i1, dev_spinor * i2){
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    (*(i1+i)).x = (*(i1+i)).x + (*(i2+i)).x;
    (*(i1+i)).y = (*(i1+i)).y + (*(i2+i)).y;
    (*(i1+i)).z = (*(i1+i)).z + (*(i2+i)).z;
    (*(i1+i)).w = (*(i1+i)).w + (*(i2+i)).w;
  }
}



__device__ inline void dev_sub_spinor_assign(dev_spinor * i1, dev_spinor * i2){
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    (*(i1+i)).x = (*(i1+i)).x - (*(i2+i)).x;
    (*(i1+i)).y = (*(i1+i)).y - (*(i2+i)).y;
    (*(i1+i)).z = (*(i1+i)).z - (*(i2+i)).z;
    (*(i1+i)).w = (*(i1+i)).w - (*(i2+i)).w;
  }
}





//multipliziert su3-Matrix mal Spinor im Dirac-Raum
//code in su3_MtV.txt -- generated with codegen
__device__ void dev_su3MtV_spintex(dev_su3 M, int pos, dev_spinor * out){

dev_spinor s1, s2;

s1 = tex1Dfetch(spin_tex,6*pos);
s2 = tex1Dfetch(spin_tex,6*pos+1);


(*(out+0)).x =  ( M[0][0].re*s1.x - M[0][0].im*s1.y ) + ( M[0][1].re*s1.z - M[0][1].im*s1.w ) + ( M[0][2].re*s2.x - M[0][2].im*s2.y );
(*(out+0)).y = ( M[0][0].re*s1.y + M[0][0].im*s1.x ) + ( M[0][1].re*s1.w + M[0][1].im*s1.z ) + ( M[0][2].re*s2.y + M[0][2].im*s2.x );


/* checked by look of eye */
(*(out+0)).z =  ( M[1][0].re*s1.x - M[1][0].im*s1.y ) + ( M[1][1].re*s1.z - M[1][1].im*s1.w ) + ( M[1][2].re*s2.x - M[1][2].im*s2.y );
/* checked */

(*(out+0)).w =  ( M[1][0].re*s1.y + M[1][0].im*s1.x ) + ( M[1][1].re*s1.w + M[1][1].im*s1.z ) + ( M[1][2].re*s2.y + M[1][2].im*s2.x );


(*(out+1)).x = ( M[2][0].re*s1.x - M[2][0].im*s1.y ) + ( M[2][1].re*s1.z - M[2][1].im*s1.w ) + ( M[2][2].re*s2.x - M[2][2].im*s2.y );
(*(out+1)).y =  ( M[2][0].re*s1.y + M[2][0].im*s1.x ) + ( M[2][1].re*s1.w + M[2][1].im*s1.z ) + ( M[2][2].re*s2.y + M[2][2].im*s2.x );



s1 = tex1Dfetch(spin_tex,6*pos+2);
(*(out+1)).z =  ( M[0][0].re*s2.z - M[0][0].im*s2.w ) + ( M[0][1].re*s1.x - M[0][1].im*s1.y ) + ( M[0][2].re*s1.z - M[0][2].im*s1.w );
(*(out+1)).w =  ( M[0][0].re*s2.w + M[0][0].im*s2.z ) + ( M[0][1].re*s1.y + M[0][1].im*s1.x ) + ( M[0][2].re*s1.w + M[0][2].im*s1.z );


(*(out+2)).x = ( M[1][0].re*s2.z - M[1][0].im*s2.w ) + ( M[1][1].re*s1.x - M[1][1].im*s1.y ) + ( M[1][2].re*s1.z - M[1][2].im*s1.w );
(*(out+2)).y =  ( M[1][0].re*s2.w + M[1][0].im*s2.z ) + ( M[1][1].re*s1.y + M[1][1].im*s1.x ) + ( M[1][2].re*s1.w + M[1][2].im*s1.z );


(*(out+2)).z =  ( M[2][0].re*s2.z - M[2][0].im*s2.w ) + ( M[2][1].re*s1.x - M[2][1].im*s1.y ) + ( M[2][2].re*s1.z - M[2][2].im*s1.w );
(*(out+2)).w =  ( M[2][0].re*s2.w + M[2][0].im*s2.z ) + ( M[2][1].re*s1.y + M[2][1].im*s1.x ) + ( M[2][2].re*s1.w + M[2][2].im*s1.z );



s1 = tex1Dfetch(spin_tex,6*pos+3);
s2 = tex1Dfetch(spin_tex,6*pos+4);
(*(out+3)).x =  ( M[0][0].re*s1.x - M[0][0].im*s1.y ) + ( M[0][1].re*s1.z - M[0][1].im*s1.w ) + ( M[0][2].re*s2.x - M[0][2].im*s2.y );
(*(out+3)).y =   ( M[0][0].re*s1.y + M[0][0].im*s1.x ) + ( M[0][1].re*s1.w + M[0][1].im*s1.z ) + ( M[0][2].re*s2.y + M[0][2].im*s2.x );


(*(out+3)).z =  ( M[1][0].re*s1.x - M[1][0].im*s1.y ) + ( M[1][1].re*s1.z - M[1][1].im*s1.w ) + ( M[1][2].re*s2.x - M[1][2].im*s2.y );
(*(out+3)).w =  ( M[1][0].re*s1.y + M[1][0].im*s1.x ) + ( M[1][1].re*s1.w + M[1][1].im*s1.z ) + ( M[1][2].re*s2.y + M[1][2].im*s2.x );


(*(out+4)).x =  ( M[2][0].re*s1.x - M[2][0].im*s1.y ) + ( M[2][1].re*s1.z - M[2][1].im*s1.w ) + ( M[2][2].re*s2.x - M[2][2].im*s2.y );
(*(out+4)).y =  ( M[2][0].re*s1.y + M[2][0].im*s1.x ) + ( M[2][1].re*s1.w + M[2][1].im*s1.z ) + ( M[2][2].re*s2.y + M[2][2].im*s2.x );



s1 = tex1Dfetch(spin_tex,6*pos+5);
(*(out+4)).z =  ( M[0][0].re*s2.z - M[0][0].im*s2.w ) + ( M[0][1].re*s1.x - M[0][1].im*s1.y ) + ( M[0][2].re*s1.z - M[0][2].im*s1.w );
(*(out+4)).w =   ( M[0][0].re*s2.w + M[0][0].im*s2.z ) + ( M[0][1].re*s1.y + M[0][1].im*s1.x ) + ( M[0][2].re*s1.w + M[0][2].im*s1.z );


(*(out+5)).x = ( M[1][0].re*s2.z - M[1][0].im*s2.w ) + ( M[1][1].re*s1.x - M[1][1].im*s1.y ) + ( M[1][2].re*s1.z - M[1][2].im*s1.w );
(*(out+5)).y =  ( M[1][0].re*s2.w + M[1][0].im*s2.z ) + ( M[1][1].re*s1.y + M[1][1].im*s1.x ) + ( M[1][2].re*s1.w + M[1][2].im*s1.z );


(*(out+5)).z =  ( M[2][0].re*s2.z - M[2][0].im*s2.w ) + ( M[2][1].re*s1.x - M[2][1].im*s1.y ) + ( M[2][2].re*s1.z - M[2][2].im*s1.w );
(*(out+5)).w =  ( M[2][0].re*s2.w + M[2][0].im*s2.z ) + ( M[2][1].re*s1.y + M[2][1].im*s1.x ) + ( M[2][2].re*s1.w + M[2][2].im*s1.z );


}










//multipliziert su3-Matrix mal Spinor im Dirac-Raum
//code in su3_MtV.txt -- generated with codegen
__device__ void dev_su3MtV(dev_su3 M, dev_spinor * s, dev_spinor * out){

(*(out+0)).x =  ( M[0][0].re*(*(s+0)).x - M[0][0].im*(*(s+0)).y ) + ( M[0][1].re*(*(s+0)).z - M[0][1].im*(*(s+0)).w ) + ( M[0][2].re*(*(s+1)).x - M[0][2].im*(*(s+1)).y );
(*(out+0)).y = ( M[0][0].re*(*(s+0)).y + M[0][0].im*(*(s+0)).x ) + ( M[0][1].re*(*(s+0)).w + M[0][1].im*(*(s+0)).z ) + ( M[0][2].re*(*(s+1)).y + M[0][2].im*(*(s+1)).x );


(*(out+0)).z =  ( M[1][0].re*(*(s+0)).x - M[1][0].im*(*(s+0)).y ) + ( M[1][1].re*(*(s+0)).z - M[1][1].im*(*(s+0)).w ) + ( M[1][2].re*(*(s+1)).x - M[1][2].im*(*(s+1)).y );
(*(out+0)).w =  ( M[1][0].re*(*(s+0)).y + M[1][0].im*(*(s+0)).x ) + ( M[1][1].re*(*(s+0)).w + M[1][1].im*(*(s+0)).z ) + ( M[1][2].re*(*(s+1)).y + M[1][2].im*(*(s+1)).x );


(*(out+1)).x = ( M[2][0].re*(*(s+0)).x - M[2][0].im*(*(s+0)).y ) + ( M[2][1].re*(*(s+0)).z - M[2][1].im*(*(s+0)).w ) + ( M[2][2].re*(*(s+1)).x - M[2][2].im*(*(s+1)).y );
(*(out+1)).y =  ( M[2][0].re*(*(s+0)).y + M[2][0].im*(*(s+0)).x ) + ( M[2][1].re*(*(s+0)).w + M[2][1].im*(*(s+0)).z ) + ( M[2][2].re*(*(s+1)).y + M[2][2].im*(*(s+1)).x );


(*(out+1)).z =  ( M[0][0].re*(*(s+1)).z - M[0][0].im*(*(s+1)).w ) + ( M[0][1].re*(*(s+2)).x - M[0][1].im*(*(s+2)).y ) + ( M[0][2].re*(*(s+2)).z - M[0][2].im*(*(s+2)).w );
(*(out+1)).w =  ( M[0][0].re*(*(s+1)).w + M[0][0].im*(*(s+1)).z ) + ( M[0][1].re*(*(s+2)).y + M[0][1].im*(*(s+2)).x ) + ( M[0][2].re*(*(s+2)).w + M[0][2].im*(*(s+2)).z );


(*(out+2)).x = ( M[1][0].re*(*(s+1)).z - M[1][0].im*(*(s+1)).w ) + ( M[1][1].re*(*(s+2)).x - M[1][1].im*(*(s+2)).y ) + ( M[1][2].re*(*(s+2)).z - M[1][2].im*(*(s+2)).w );
(*(out+2)).y =  ( M[1][0].re*(*(s+1)).w + M[1][0].im*(*(s+1)).z ) + ( M[1][1].re*(*(s+2)).y + M[1][1].im*(*(s+2)).x ) + ( M[1][2].re*(*(s+2)).w + M[1][2].im*(*(s+2)).z );


(*(out+2)).z =  ( M[2][0].re*(*(s+1)).z - M[2][0].im*(*(s+1)).w ) + ( M[2][1].re*(*(s+2)).x - M[2][1].im*(*(s+2)).y ) + ( M[2][2].re*(*(s+2)).z - M[2][2].im*(*(s+2)).w );
(*(out+2)).w =  ( M[2][0].re*(*(s+1)).w + M[2][0].im*(*(s+1)).z ) + ( M[2][1].re*(*(s+2)).y + M[2][1].im*(*(s+2)).x ) + ( M[2][2].re*(*(s+2)).w + M[2][2].im*(*(s+2)).z );


(*(out+3)).x =  ( M[0][0].re*(*(s+3)).x - M[0][0].im*(*(s+3)).y ) + ( M[0][1].re*(*(s+3)).z - M[0][1].im*(*(s+3)).w ) + ( M[0][2].re*(*(s+4)).x - M[0][2].im*(*(s+4)).y );
(*(out+3)).y =   ( M[0][0].re*(*(s+3)).y + M[0][0].im*(*(s+3)).x ) + ( M[0][1].re*(*(s+3)).w + M[0][1].im*(*(s+3)).z ) + ( M[0][2].re*(*(s+4)).y + M[0][2].im*(*(s+4)).x );


(*(out+3)).z =  ( M[1][0].re*(*(s+3)).x - M[1][0].im*(*(s+3)).y ) + ( M[1][1].re*(*(s+3)).z - M[1][1].im*(*(s+3)).w ) + ( M[1][2].re*(*(s+4)).x - M[1][2].im*(*(s+4)).y );
(*(out+3)).w =  ( M[1][0].re*(*(s+3)).y + M[1][0].im*(*(s+3)).x ) + ( M[1][1].re*(*(s+3)).w + M[1][1].im*(*(s+3)).z ) + ( M[1][2].re*(*(s+4)).y + M[1][2].im*(*(s+4)).x );


(*(out+4)).x =  ( M[2][0].re*(*(s+3)).x - M[2][0].im*(*(s+3)).y ) + ( M[2][1].re*(*(s+3)).z - M[2][1].im*(*(s+3)).w ) + ( M[2][2].re*(*(s+4)).x - M[2][2].im*(*(s+4)).y );
(*(out+4)).y =  ( M[2][0].re*(*(s+3)).y + M[2][0].im*(*(s+3)).x ) + ( M[2][1].re*(*(s+3)).w + M[2][1].im*(*(s+3)).z ) + ( M[2][2].re*(*(s+4)).y + M[2][2].im*(*(s+4)).x );


(*(out+4)).z =  ( M[0][0].re*(*(s+4)).z - M[0][0].im*(*(s+4)).w ) + ( M[0][1].re*(*(s+5)).x - M[0][1].im*(*(s+5)).y ) + ( M[0][2].re*(*(s+5)).z - M[0][2].im*(*(s+5)).w );
(*(out+4)).w =   ( M[0][0].re*(*(s+4)).w + M[0][0].im*(*(s+4)).z ) + ( M[0][1].re*(*(s+5)).y + M[0][1].im*(*(s+5)).x ) + ( M[0][2].re*(*(s+5)).w + M[0][2].im*(*(s+5)).z );


(*(out+5)).x = ( M[1][0].re*(*(s+4)).z - M[1][0].im*(*(s+4)).w ) + ( M[1][1].re*(*(s+5)).x - M[1][1].im*(*(s+5)).y ) + ( M[1][2].re*(*(s+5)).z - M[1][2].im*(*(s+5)).w );
(*(out+5)).y =  ( M[1][0].re*(*(s+4)).w + M[1][0].im*(*(s+4)).z ) + ( M[1][1].re*(*(s+5)).y + M[1][1].im*(*(s+5)).x ) + ( M[1][2].re*(*(s+5)).w + M[1][2].im*(*(s+5)).z );


(*(out+5)).z =  ( M[2][0].re*(*(s+4)).z - M[2][0].im*(*(s+4)).w ) + ( M[2][1].re*(*(s+5)).x - M[2][1].im*(*(s+5)).y ) + ( M[2][2].re*(*(s+5)).z - M[2][2].im*(*(s+5)).w );
(*(out+5)).w =  ( M[2][0].re*(*(s+4)).w + M[2][0].im*(*(s+4)).z ) + ( M[2][1].re*(*(s+5)).y + M[2][1].im*(*(s+5)).x ) + ( M[2][2].re*(*(s+5)).w + M[2][2].im*(*(s+5)).z );
}





//multipliziert gedaggerte su3-Matrix mal Spinor im Dirac-Raum  -- generated with codegen
__device__ void dev_su3MdaggertV(dev_su3 M, dev_spinor * s, dev_spinor * out){
  dev_complex help1;
help1.re = M[0][0].re*(*(s+0)).x + M[0][0].im*(*(s+0)).y + M[1][0].re*(*(s+0)).z + M[1][0].im*(*(s+0)).w + M[2][0].re*(*(s+1)).x + M[2][0].im*(*(s+1)).y;
(*(out+0)).x = help1.re;
help1.im = M[0][0].re*(*(s+0)).y - M[0][0].im*(*(s+0)).x + M[1][0].re*(*(s+0)).w - M[1][0].im*(*(s+0)).z + M[2][0].re*(*(s+1)).y - M[2][0].im*(*(s+1)).x;
(*(out+0)).y = help1.im;

help1.re = M[0][1].re*(*(s+0)).x + M[0][1].im*(*(s+0)).y + M[1][1].re*(*(s+0)).z + M[1][1].im*(*(s+0)).w + M[2][1].re*(*(s+1)).x + M[2][1].im*(*(s+1)).y;
(*(out+0)).z = help1.re;
help1.im = M[0][1].re*(*(s+0)).y - M[0][1].im*(*(s+0)).x + M[1][1].re*(*(s+0)).w - M[1][1].im*(*(s+0)).z + M[2][1].re*(*(s+1)).y - M[2][1].im*(*(s+1)).x;
(*(out+0)).w = help1.im;

help1.re = M[0][2].re*(*(s+0)).x + M[0][2].im*(*(s+0)).y + M[1][2].re*(*(s+0)).z + M[1][2].im*(*(s+0)).w + M[2][2].re*(*(s+1)).x + M[2][2].im*(*(s+1)).y;
(*(out+1)).x = help1.re;
help1.im = M[0][2].re*(*(s+0)).y - M[0][2].im*(*(s+0)).x + M[1][2].re*(*(s+0)).w - M[1][2].im*(*(s+0)).z + M[2][2].re*(*(s+1)).y - M[2][2].im*(*(s+1)).x;
(*(out+1)).y = help1.im;

help1.re = M[0][0].re*(*(s+1)).z + M[0][0].im*(*(s+1)).w + M[1][0].re*(*(s+2)).x + M[1][0].im*(*(s+2)).y + M[2][0].re*(*(s+2)).z + M[2][0].im*(*(s+2)).w;
(*(out+1)).z = help1.re;
help1.im = M[0][0].re*(*(s+1)).w - M[0][0].im*(*(s+1)).z + M[1][0].re*(*(s+2)).y - M[1][0].im*(*(s+2)).x + M[2][0].re*(*(s+2)).w - M[2][0].im*(*(s+2)).z;
(*(out+1)).w = help1.im;

help1.re = M[0][1].re*(*(s+1)).z + M[0][1].im*(*(s+1)).w + M[1][1].re*(*(s+2)).x + M[1][1].im*(*(s+2)).y + M[2][1].re*(*(s+2)).z + M[2][1].im*(*(s+2)).w;
(*(out+2)).x = help1.re;
help1.im = M[0][1].re*(*(s+1)).w - M[0][1].im*(*(s+1)).z + M[1][1].re*(*(s+2)).y - M[1][1].im*(*(s+2)).x + M[2][1].re*(*(s+2)).w - M[2][1].im*(*(s+2)).z;
(*(out+2)).y = help1.im;

help1.re = M[0][2].re*(*(s+1)).z + M[0][2].im*(*(s+1)).w + M[1][2].re*(*(s+2)).x + M[1][2].im*(*(s+2)).y + M[2][2].re*(*(s+2)).z + M[2][2].im*(*(s+2)).w;
(*(out+2)).z = help1.re;
help1.im = M[0][2].re*(*(s+1)).w - M[0][2].im*(*(s+1)).z + M[1][2].re*(*(s+2)).y - M[1][2].im*(*(s+2)).x + M[2][2].re*(*(s+2)).w - M[2][2].im*(*(s+2)).z;
(*(out+2)).w = help1.im;

help1.re = M[0][0].re*(*(s+3)).x + M[0][0].im*(*(s+3)).y + M[1][0].re*(*(s+3)).z + M[1][0].im*(*(s+3)).w + M[2][0].re*(*(s+4)).x + M[2][0].im*(*(s+4)).y;
(*(out+3)).x = help1.re;
help1.im = M[0][0].re*(*(s+3)).y - M[0][0].im*(*(s+3)).x + M[1][0].re*(*(s+3)).w - M[1][0].im*(*(s+3)).z + M[2][0].re*(*(s+4)).y - M[2][0].im*(*(s+4)).x;
(*(out+3)).y = help1.im;

help1.re = M[0][1].re*(*(s+3)).x + M[0][1].im*(*(s+3)).y + M[1][1].re*(*(s+3)).z + M[1][1].im*(*(s+3)).w + M[2][1].re*(*(s+4)).x + M[2][1].im*(*(s+4)).y;
(*(out+3)).z = help1.re;
help1.im = M[0][1].re*(*(s+3)).y - M[0][1].im*(*(s+3)).x + M[1][1].re*(*(s+3)).w - M[1][1].im*(*(s+3)).z + M[2][1].re*(*(s+4)).y - M[2][1].im*(*(s+4)).x;
(*(out+3)).w = help1.im;

help1.re = M[0][2].re*(*(s+3)).x + M[0][2].im*(*(s+3)).y + M[1][2].re*(*(s+3)).z + M[1][2].im*(*(s+3)).w + M[2][2].re*(*(s+4)).x + M[2][2].im*(*(s+4)).y;
(*(out+4)).x = help1.re;
help1.im = M[0][2].re*(*(s+3)).y - M[0][2].im*(*(s+3)).x + M[1][2].re*(*(s+3)).w - M[1][2].im*(*(s+3)).z + M[2][2].re*(*(s+4)).y - M[2][2].im*(*(s+4)).x;
(*(out+4)).y = help1.im;

help1.re = M[0][0].re*(*(s+4)).z + M[0][0].im*(*(s+4)).w + M[1][0].re*(*(s+5)).x + M[1][0].im*(*(s+5)).y + M[2][0].re*(*(s+5)).z + M[2][0].im*(*(s+5)).w;
(*(out+4)).z = help1.re;
help1.im = M[0][0].re*(*(s+4)).w - M[0][0].im*(*(s+4)).z + M[1][0].re*(*(s+5)).y - M[1][0].im*(*(s+5)).x + M[2][0].re*(*(s+5)).w - M[2][0].im*(*(s+5)).z;
(*(out+4)).w = help1.im;

help1.re = M[0][1].re*(*(s+4)).z + M[0][1].im*(*(s+4)).w + M[1][1].re*(*(s+5)).x + M[1][1].im*(*(s+5)).y + M[2][1].re*(*(s+5)).z + M[2][1].im*(*(s+5)).w;
(*(out+5)).x = help1.re;
help1.im = M[0][1].re*(*(s+4)).w - M[0][1].im*(*(s+4)).z + M[1][1].re*(*(s+5)).y - M[1][1].im*(*(s+5)).x + M[2][1].re*(*(s+5)).w - M[2][1].im*(*(s+5)).z;
(*(out+5)).y = help1.im;

help1.re = M[0][2].re*(*(s+4)).z + M[0][2].im*(*(s+4)).w + M[1][2].re*(*(s+5)).x + M[1][2].im*(*(s+5)).y + M[2][2].re*(*(s+5)).z + M[2][2].im*(*(s+5)).w;
(*(out+5)).z = help1.re;
help1.im = M[0][2].re*(*(s+4)).w - M[0][2].im*(*(s+4)).z + M[1][2].re*(*(s+5)).y - M[1][2].im*(*(s+5)).x + M[2][2].re*(*(s+5)).w - M[2][2].im*(*(s+5)).z;
(*(out+5)).w = help1.im;
}




// Gamma t
__device__ void dev_Gamma0(dev_spinor * in){
  REAL tempre,tempim;
     tempre = (*(in+0)).x;
     tempim = (*(in+0)).y;
     (*(in+0)).x = -1.0*(*(in+3)).x;
     (*(in+0)).y = -1.0*(*(in+3)).y;
     (*(in+3)).x = -1.0*tempre;
     (*(in+3)).y = -1.0*tempim;     
     
     tempre = (*(in+0)).z;
     tempim = (*(in+0)).w;
     (*(in+0)).z = -1.0*(*(in+3)).z;
     (*(in+0)).w = -1.0*(*(in+3)).w;
     (*(in+3)).z = -1.0*tempre;
     (*(in+3)).w = -1.0*tempim; 
 
 
     tempre = (*(in+1)).x;
     tempim = (*(in+1)).y;
     (*(in+1)).x = -1.0*(*(in+4)).x;
     (*(in+1)).y = -1.0*(*(in+4)).y;
     (*(in+4)).x = -1.0*tempre;
     (*(in+4)).y = -1.0*tempim;     
     
     tempre = (*(in+1)).z;
     tempim = (*(in+1)).w;
     (*(in+1)).z = -1.0*(*(in+4)).z;
     (*(in+1)).w = -1.0*(*(in+4)).w;
     (*(in+4)).z = -1.0*tempre;
     (*(in+4)).w = -1.0*tempim;     
     
     
     tempre = (*(in+2)).x;
     tempim = (*(in+2)).y;
     (*(in+2)).x = -1.0*(*(in+5)).x;
     (*(in+2)).y = -1.0*(*(in+5)).y;
     (*(in+5)).x = -1.0*tempre;
     (*(in+5)).y = -1.0*tempim;     
   
   
     tempre = (*(in+2)).z;
     tempim = (*(in+2)).w;
     (*(in+2)).z = -1.0*(*(in+5)).z;
     (*(in+2)).w = -1.0*(*(in+5)).w;
     (*(in+5)).z = -1.0*tempre;
     (*(in+5)).w = -1.0*tempim;
}



//Gamma z
__device__ void dev_Gamma3(dev_spinor * in){
  REAL tempre,tempim;
     tempre = (*(in+0)).x;
     tempim = (*(in+0)).y;
     (*(in+0)).x = (*(in+3)).y;
     (*(in+0)).y = -1.0*(*(in+3)).x;
     (*(in+3)).x = -1.0*tempim;
     (*(in+3)).y = tempre;    
     
     tempre = (*(in+0)).z;
     tempim = (*(in+0)).w;
     (*(in+0)).z = (*(in+3)).w;
     (*(in+0)).w = -1.0*(*(in+3)).z;
     (*(in+3)).z = -1.0*tempim;
     (*(in+3)).w = tempre;    
     
     
     tempre = (*(in+1)).x;
     tempim = (*(in+1)).y;
     (*(in+1)).x = (*(in+4)).y;
     (*(in+1)).y = -1.0*(*(in+4)).x;
     (*(in+4)).x = -1.0*tempim;
     (*(in+4)).y = tempre;     
     
     
     tempre = (*(in+1)).z;
     tempim = (*(in+1)).w;
     (*(in+1)).z = -1.0*(*(in+4)).w;
     (*(in+1)).w = (*(in+4)).z;
     (*(in+4)).z  = tempim;
     (*(in+4)).w  = -1.0*tempre;     
     
     
     
     tempre = (*(in+2)).x;
     tempim = (*(in+2)).y;
     (*(in+2)).x = -1.0*(*(in+5)).y;
     (*(in+2)).y = (*(in+5)).x;
     (*(in+5)).x = tempim;
     (*(in+5)).y = -1.0*tempre;    
     
     
     tempre = (*(in+2)).z;
     tempim = (*(in+2)).w;
     (*(in+2)).z = -1.0*(*(in+5)).w;
     (*(in+2)).w = (*(in+5)).z;
     (*(in+5)).z = tempim;
     (*(in+5)).w = -1.0*tempre;

}



//Gamma y
__device__ void dev_Gamma2(dev_spinor * in){
  REAL tempre,tempim;
     tempre = (*(in+0)).x;
     tempim = (*(in+0)).y;
     (*(in+0)).x = -1.0*(*(in+4)).z;
     (*(in+0)).y = -1.0*(*(in+4)).w;
     (*(in+4)).z = -1.0*tempre;
     (*(in+4)).w = -1.0*tempim;    
     
     tempre = (*(in+0)).z;
     tempim = (*(in+0)).w;
     (*(in+0)).z = -1.0*(*(in+5)).x;
     (*(in+0)).w = -1.0*(*(in+5)).y;
     (*(in+5)).x = -1.0*tempre;
     (*(in+5)).y = -1.0*tempim;     
     
     
     tempre = (*(in+1)).x;
     tempim = (*(in+1)).y;
     (*(in+1)).x = -1.0*(*(in+5)).z;
     (*(in+1)).y = -1.0*(*(in+5)).w;
     (*(in+5)).z = -1.0*tempre;
     (*(in+5)).w = -1.0*tempim;     
     
     tempre = (*(in+1)).z;
     tempim = (*(in+1)).w;
     (*(in+1)).z = (*(in+3)).x;
     (*(in+1)).w = (*(in+3)).y;
     (*(in+3)).x = tempre;
     (*(in+3)).y = tempim;     
     
     tempre = (*(in+2)).x;
     tempim = (*(in+2)).y;
     (*(in+2)).x = (*(in+3)).z;
     (*(in+2)).y = (*(in+3)).w;
     (*(in+3)).z = tempre;
     (*(in+3)).w = tempim;     
     
     
     tempre = (*(in+2)).z;
     tempim = (*(in+2)).w;
     (*(in+2)).z = (*(in+4)).x;
     (*(in+2)).w = (*(in+4)).y;
     (*(in+4)).x = tempre;
     (*(in+4)).y = tempim;
}



//Gamma x
__device__ void dev_Gamma1(dev_spinor * in){
  REAL tempre,tempim;
     tempre = (*(in+0)).x;
     tempim = (*(in+0)).y;
     (*(in+0)).x = (*(in+4)).w;
     (*(in+0)).y = -1.0*(*(in+4)).z;
     (*(in+4)).z  = -1.0*tempim;
     (*(in+4)).w  = tempre;    
     
     tempre = (*(in+0)).z;
     tempim = (*(in+0)).w;
     (*(in+0)).z = (*(in+5)).y;
     (*(in+0)).w = -1.0*(*(in+5)).x;
     (*(in+5)).x = -1.0*tempim;
     (*(in+5)).y = tempre;     
     
     tempre = (*(in+1)).x;
     tempim = (*(in+1)).y;
     (*(in+1)).x = (*(in+5)).w;
     (*(in+1)).y = -1.0*(*(in+5)).z;
     (*(in+5)).z = -1.0*tempim;
     (*(in+5)).w = tempre;     
     
     tempre = (*(in+1)).z;
     tempim = (*(in+1)).w;
     (*(in+1)).z = (*(in+3)).y;
     (*(in+1)).w = -1.0*(*(in+3)).x;
     (*(in+3)).x = -1.0*tempim;
     (*(in+3)).y = tempre;     
     
     tempre = (*(in+2)).x;
     tempim = (*(in+2)).y;
     (*(in+2)).x = (*(in+3)).w;
     (*(in+2)).y = -1.0*(*(in+3)).z;
     (*(in+3)).z = -1.0*tempim;
     (*(in+3)).w = tempre;     
     
     
     tempre = (*(in+2)).z;
     tempim = (*(in+2)).w;
     (*(in+2)).z = (*(in+4)).y;
     (*(in+2)).w = -1.0*(*(in+4)).x;
     (*(in+4)).x = -1.0*tempim;
     (*(in+4)).y = tempre;
  
}



__device__ void dev_Gamma5(dev_spinor * in){
          (*(in+3)).x = -1.0*(*(in+3)).x;
          (*(in+3)).y = -1.0*(*(in+3)).y;
          (*(in+3)).z = -1.0*(*(in+3)).z;
          (*(in+3)).w = -1.0*(*(in+3)).w;
          (*(in+4)).x = -1.0*(*(in+4)).x;
          (*(in+4)).y = -1.0*(*(in+4)).y; 

          (*(in+4)).z = -1.0*(*(in+4)).z;
          (*(in+4)).w = -1.0*(*(in+4)).w;
          (*(in+5)).x = -1.0*(*(in+5)).x;
          (*(in+5)).y = -1.0*(*(in+5)).y;
          (*(in+5)).z = -1.0*(*(in+5)).z;
          (*(in+5)).w = -1.0*(*(in+5)).w;  
}





// older version, all in one function
__device__ void dev_GammatV(int mu, dev_spinor * in){//multipliziert Gamma(mu)*V effizientes ausnutzen der Nullen 
 REAL tempre,tempim;
 /* ORDER: t, z, y, x*/
 switch (mu){
 
 case 0:
     tempre = (*(in+0)).x;
     tempim = (*(in+0)).y;
     (*(in+0)).x = -1.0*(*(in+3)).x;
     (*(in+0)).y = -1.0*(*(in+3)).y;
     (*(in+3)).x = -1.0*tempre;
     (*(in+3)).y = -1.0*tempim;     
     
     tempre = (*(in+0)).z;
     tempim = (*(in+0)).w;
     (*(in+0)).z = -1.0*(*(in+3)).z;
     (*(in+0)).w = -1.0*(*(in+3)).w;
     (*(in+3)).z = -1.0*tempre;
     (*(in+3)).w = -1.0*tempim; 
 
 
     tempre = (*(in+1)).x;
     tempim = (*(in+1)).y;
     (*(in+1)).x = -1.0*(*(in+4)).x;
     (*(in+1)).y = -1.0*(*(in+4)).y;
     (*(in+4)).x = -1.0*tempre;
     (*(in+4)).y = -1.0*tempim;     
     
     tempre = (*(in+1)).z;
     tempim = (*(in+1)).w;
     (*(in+1)).z = -1.0*(*(in+4)).z;
     (*(in+1)).w = -1.0*(*(in+4)).w;
     (*(in+4)).z = -1.0*tempre;
     (*(in+4)).w = -1.0*tempim;     
     
     
     tempre = (*(in+2)).x;
     tempim = (*(in+2)).y;
     (*(in+2)).x = -1.0*(*(in+5)).x;
     (*(in+2)).y = -1.0*(*(in+5)).y;
     (*(in+5)).x = -1.0*tempre;
     (*(in+5)).y = -1.0*tempim;     
   
   
     tempre = (*(in+2)).z;
     tempim = (*(in+2)).w;
     (*(in+2)).z = -1.0*(*(in+5)).z;
     (*(in+2)).w = -1.0*(*(in+5)).w;
     (*(in+5)).z = -1.0*tempre;
     (*(in+5)).w = -1.0*tempim;

 break;
 
 case 1:
     tempre = (*(in+0)).x;
     tempim = (*(in+0)).y;
     (*(in+0)).x = (*(in+3)).y;
     (*(in+0)).y = -1.0*(*(in+3)).x;
     (*(in+3)).x = -1.0*tempim;
     (*(in+3)).y = tempre;    
     
     tempre = (*(in+0)).z;
     tempim = (*(in+0)).w;
     (*(in+0)).z = (*(in+3)).w;
     (*(in+0)).w = -1.0*(*(in+3)).z;
     (*(in+3)).z = -1.0*tempim;
     (*(in+3)).w = tempre;    
     
     
     tempre = (*(in+1)).x;
     tempim = (*(in+1)).y;
     (*(in+1)).x = (*(in+4)).y;
     (*(in+1)).y = -1.0*(*(in+4)).x;
     (*(in+4)).x = -1.0*tempim;
     (*(in+4)).y = tempre;     
     
     
     tempre = (*(in+1)).z;
     tempim = (*(in+1)).w;
     (*(in+1)).z = -1.0*(*(in+4)).w;
     (*(in+1)).w = (*(in+4)).z;
     (*(in+4)).z  = tempim;
     (*(in+4)).w  = -1.0*tempre;     
     
     
     
     tempre = (*(in+2)).x;
     tempim = (*(in+2)).y;
     (*(in+2)).x = -1.0*(*(in+5)).y;
     (*(in+2)).y = (*(in+5)).x;
     (*(in+5)).x = tempim;
     (*(in+5)).y = -1.0*tempre;    
     
     
     tempre = (*(in+2)).z;
     tempim = (*(in+2)).w;
     (*(in+2)).z = -1.0*(*(in+5)).w;
     (*(in+2)).w = (*(in+5)).z;
     (*(in+5)).z = tempim;
     (*(in+5)).w = -1.0*tempre;


 break;
 
 case 2:
     tempre = (*(in+0)).x;
     tempim = (*(in+0)).y;
     (*(in+0)).x = -1.0*(*(in+4)).z;
     (*(in+0)).y = -1.0*(*(in+4)).w;
     (*(in+4)).z = -1.0*tempre;
     (*(in+4)).w = -1.0*tempim;    
     
     tempre = (*(in+0)).z;
     tempim = (*(in+0)).w;
     (*(in+0)).z = -1.0*(*(in+5)).x;
     (*(in+0)).w = -1.0*(*(in+5)).y;
     (*(in+5)).x = -1.0*tempre;
     (*(in+5)).y = -1.0*tempim;     
     
     
     tempre = (*(in+1)).x;
     tempim = (*(in+1)).y;
     (*(in+1)).x = -1.0*(*(in+5)).z;
     (*(in+1)).y = -1.0*(*(in+5)).w;
     (*(in+5)).z = -1.0*tempre;
     (*(in+5)).w = -1.0*tempim;     
     
     tempre = (*(in+1)).z;
     tempim = (*(in+1)).w;
     (*(in+1)).z = (*(in+3)).x;
     (*(in+1)).w = (*(in+3)).y;
     (*(in+3)).x = tempre;
     (*(in+3)).y = tempim;     
     
     tempre = (*(in+2)).x;
     tempim = (*(in+2)).y;
     (*(in+2)).x = (*(in+3)).z;
     (*(in+2)).y = (*(in+3)).w;
     (*(in+3)).z = tempre;
     (*(in+3)).w = tempim;     
     
     
     tempre = (*(in+2)).z;
     tempim = (*(in+2)).w;
     (*(in+2)).z = (*(in+4)).x;
     (*(in+2)).w = (*(in+4)).y;
     (*(in+4)).x = tempre;
     (*(in+4)).y = tempim;

 break; 
 
 case 3:


     tempre = (*(in+0)).x;
     tempim = (*(in+0)).y;
     (*(in+0)).x = (*(in+4)).w;
     (*(in+0)).y = -1.0*(*(in+4)).z;
     (*(in+4)).z  = -1.0*tempim;
     (*(in+4)).w  = tempre;    
     
     tempre = (*(in+0)).z;
     tempim = (*(in+0)).w;
     (*(in+0)).z = (*(in+5)).y;
     (*(in+0)).w = -1.0*(*(in+5)).x;
     (*(in+5)).x = -1.0*tempim;
     (*(in+5)).y = tempre;     
     
     tempre = (*(in+1)).x;
     tempim = (*(in+1)).y;
     (*(in+1)).x = (*(in+5)).w;
     (*(in+1)).y = -1.0*(*(in+5)).z;
     (*(in+5)).z = -1.0*tempim;
     (*(in+5)).w = tempre;     
     
     tempre = (*(in+1)).z;
     tempim = (*(in+1)).w;
     (*(in+1)).z = (*(in+3)).y;
     (*(in+1)).w = -1.0*(*(in+3)).x;
     (*(in+3)).x = -1.0*tempim;
     (*(in+3)).y = tempre;     
     
     tempre = (*(in+2)).x;
     tempim = (*(in+2)).y;
     (*(in+2)).x = (*(in+3)).w;
     (*(in+2)).y = -1.0*(*(in+3)).z;
     (*(in+3)).z = -1.0*tempim;
     (*(in+3)).w = tempre;     
     
     
     tempre = (*(in+2)).z;
     tempim = (*(in+2)).w;
     (*(in+2)).z = (*(in+4)).y;
     (*(in+2)).w = -1.0*(*(in+4)).x;
     (*(in+4)).x = -1.0*tempim;
     (*(in+4)).y = tempre;
     
     
 break;
 
 
 case 4:
  
          (*(in+3)).x = -1.0*(*(in+3)).x;
          (*(in+3)).y = -1.0*(*(in+3)).y;
          (*(in+3)).z = -1.0*(*(in+3)).z;
          (*(in+3)).w = -1.0*(*(in+3)).w;
          (*(in+4)).x = -1.0*(*(in+4)).x;
          (*(in+4)).y = -1.0*(*(in+4)).y; 

          (*(in+4)).z = -1.0*(*(in+4)).z;
          (*(in+4)).w = -1.0*(*(in+4)).w;
          (*(in+5)).x = -1.0*(*(in+5)).x;
          (*(in+5)).y = -1.0*(*(in+5)).y;
          (*(in+5)).z = -1.0*(*(in+5)).z;
          (*(in+5)).w = -1.0*(*(in+5)).w;  
 break;
 }
}




// reconstruction of the link fields from two rows of the su3 matrix
// numbers are fetched from texture cache
__device__ void dev_reconstructgf_2vtexref (int pos, dev_su3* gf){
  dev_complex help1;
  dev_complex help2;
  float4 gfin;
  
  gfin = tex1Dfetch(gf_tex,3*pos);
  //first row
  (*gf)[0][0].re = gfin.x;
  (*gf)[0][0].im = gfin.y;
  (*gf)[0][1].re = gfin.z;
  (*gf)[0][1].im = gfin.w;
  gfin = tex1Dfetch(gf_tex,3*pos+1);
  (*gf)[0][2].re = gfin.x;
  (*gf)[0][2].im = gfin.y;
  //second row
  (*gf)[1][0].re = gfin.z;
  (*gf)[1][0].im = gfin.w;
  gfin = tex1Dfetch(gf_tex,3*pos+2);
  (*gf)[1][1].re = gfin.x;
  (*gf)[1][1].im = gfin.y;
  (*gf)[1][2].re = gfin.z;
  (*gf)[1][2].im = gfin.w;
  
  //third row from cconj(cross product of first and second row)
  help1 = dev_cmult((*gf)[0][1],(*gf)[1][2]);
  help2 = dev_cmult((*gf)[0][2],(*gf)[1][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[2][0] = help1;

  
  help1 = dev_cmult((*gf)[0][2],(*gf)[1][0]);
  help2 = dev_cmult((*gf)[0][0],(*gf)[1][2]);
  help1 = dev_cconj(dev_csub(help1,help2));
  
  (*gf)[2][1] = help1;
  
  help1 = dev_cmult((*gf)[0][0],(*gf)[1][1]);
  help2 = dev_cmult((*gf)[0][1],(*gf)[1][0]);
  help1 = dev_cconj(dev_csub(help1,help2));
  
  (*gf)[2][2] = help1;
  
  return;
}




// su3 - dagger reconstruction from two rows  
__device__ void dev_reconstructgf_2vtexref_dagger (int pos, dev_su3* gf){
  dev_complex help1;
  dev_complex help2;
  float4 gfin;
  
  
  //first column (minus in im for complex conj.)
  gfin = tex1Dfetch(gf_tex,3*pos);
  (*gf)[0][0].re = gfin.x;
  (*gf)[0][0].im = -gfin.y;
  (*gf)[1][0].re = gfin.z;
  (*gf)[1][0].im = -gfin.w;
  gfin = tex1Dfetch(gf_tex,3*pos+1);
  (*gf)[2][0].re = gfin.x;
  (*gf)[2][0].im = -gfin.y;
  
  //second  column (minus in im for complex conj.)
  (*gf)[0][1].re = gfin.z;
  (*gf)[0][1].im = -gfin.w;
  gfin = tex1Dfetch(gf_tex,3*pos+2);
  (*gf)[1][1].re = gfin.x;
  (*gf)[1][1].im = -gfin.y;
  (*gf)[2][1].re = gfin.z;
  (*gf)[2][1].im = -gfin.w;
  
  //third column from (cross product of cconj(first column) and cconj(second column))
 
  help1 = dev_cconj(dev_cmult((*gf)[1][0],(*gf)[2][1]));
  help2 = dev_cconj(dev_cmult((*gf)[2][0],(*gf)[1][1]));
  help1 = dev_csub(help1,help2);
  (*gf)[0][2] = help1;

  
  help1 = dev_cconj(dev_cmult((*gf)[2][0],(*gf)[0][1]));
  help2 = dev_cconj(dev_cmult((*gf)[0][0],(*gf)[2][1]));
  help1 = dev_csub(help1,help2);
  (*gf)[1][2] = help1;

  
  help1 = dev_cconj(dev_cmult((*gf)[0][0],(*gf)[1][1]));
  help2 = dev_cconj(dev_cmult((*gf)[1][0],(*gf)[0][1]));
  help1 = dev_csub(help1,help2);
  (*gf)[2][2] = help1;
  
  
  /* does this also work?
  help1 = dev_cmult((*gf)[1][0],(*gf)[2][1]);
  help2 = dev_cmult((*gf)[2][0],(*gf)[1][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[0][2] = help1;

  
  help1 = dev_cmult((*gf)[2][0],(*gf)[0][1]);
  help2 = dev_cmult((*gf)[0][0],(*gf)[2][1]);
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[1][2] = help1;

  
  help1 = dev_cmult((*gf)[0][0],(*gf)[1][1]);
  help2 = dev_cmult((*gf)[1][0],(*gf)[0][1]));
  help1 = dev_cconj(dev_csub(help1,help2));
  (*gf)[2][2] = help1;
  */
}








__device__ void dev_reconstructgf_8texref (int pos, dev_su3* gf){

  float4 gfin;
  REAL N, one_over_N, help;
  dev_complex p1,p2, chelp1, chelp2;
  
  gfin = tex1Dfetch(gf_tex,2*pos);
  // read a2 a3
  (*gf)[0][1].re = gfin.x;
  (*gf)[0][1].im = gfin.y;
  (*gf)[0][2].re = gfin.z;
  (*gf)[0][2].im = gfin.w;  
 
  help = gfin.x*gfin.x + gfin.y*gfin.y + gfin.z*gfin.z + gfin.w*gfin.w; // use later on
  N = sqrtf(help);
  one_over_N = 1.0/N;
  
  // read theta_a1, theta_c1, b1
  gfin = tex1Dfetch(gf_tex,2*pos + 1);
  
  // reconstruct a1
  help = sqrtf(1.0f - help);
  (*gf)[0][0].re = help*cosf(gfin.x);
  (*gf)[0][0].im = help*sinf(gfin.x);
  
  // assign b1
  (*gf)[1][0].re = gfin.z;
  (*gf)[1][0].im = gfin.w;
  
  // p2 = 1/N b1
  p2.re = one_over_N*(*gf)[1][0].re;
  p2.im = one_over_N*(*gf)[1][0].im;  


  // reconstruct c1
  help = sqrtf(1.0 - 
              (*gf)[0][0].re * (*gf)[0][0].re - (*gf)[0][0].im * (*gf)[0][0].im - 
              (*gf)[1][0].re * (*gf)[1][0].re - (*gf)[1][0].im * (*gf)[1][0].im
          );
  (*gf)[2][0].re = help*cosf(gfin.y);
  (*gf)[2][0].im = help*sinf(gfin.y);

  
  // p1 = 1/N*cconj(c1)
  p1.re = one_over_N*(*gf)[2][0].re;
  p1.im = - one_over_N*(*gf)[2][0].im;
  
  
  // calculate b2
  chelp1 = dev_cmult(p1,  dev_cconj( (*gf)[0][2] )   );
  chelp2 = dev_cmult(p2, dev_cmult( dev_cconj((*gf)[0][0]) , (*gf)[0][1] )  );
  chelp1 = dev_cadd(chelp1, chelp2);
  (*gf)[1][1] = dev_crealmult(chelp1, -one_over_N);
  
  // calculate b3
  chelp1 = dev_cmult(p1,  dev_cconj( (*gf)[0][1] )   );
  chelp2 = dev_cmult(p2, dev_cmult( dev_cconj((*gf)[0][0]) , (*gf)[0][2] )  );
  chelp1 = dev_cadd(chelp1, chelp2);
  (*gf)[1][2] = dev_crealmult(chelp1, -one_over_N);
  
  // calculate c2
  chelp1 = dev_cmult(  dev_cconj(p2) ,  dev_cconj( (*gf)[0][2] )   );
  chelp2 = dev_cmult(  dev_cconj(p1) , 
                       dev_cmult(  dev_cconj( (*gf)[0][0] ) , (*gf)[0][1] )
                     );
  chelp1 = dev_csub(chelp1, chelp2);
  (*gf)[2][1] = dev_crealmult(chelp1, one_over_N);
  
  
  // calculate c3
  chelp1 = dev_cmult(  dev_cconj(p2) ,  dev_cconj( (*gf)[0][1] )   );
  chelp2 = dev_cmult(  dev_cconj(p1) , 
                       dev_cmult(  dev_cconj( (*gf)[0][0] ) , (*gf)[0][2] )
                     );
  chelp1 = dev_csub(chelp1, chelp2);
  (*gf)[2][2] = dev_crealmult(chelp1, one_over_N);
                       
}



__device__ void dev_reconstructgf_8texref_dagger (int pos, dev_su3* gf){


  float4 gfin;
  REAL N, one_over_N, help;
  dev_complex p1,p2, chelp1, chelp2;
  
  gfin = tex1Dfetch(gf_tex,2*pos);
  // read a2 a3
  (*gf)[0][1].re = gfin.x;
  (*gf)[0][1].im = gfin.y;
  (*gf)[0][2].re = gfin.z;
  (*gf)[0][2].im = gfin.w;  
 
  help = gfin.x*gfin.x + gfin.y*gfin.y + gfin.z*gfin.z + gfin.w*gfin.w; // use later on
  N = sqrtf(help);
  one_over_N = 1.0/N;
  
  // read theta_a1, theta_c1, b1
  gfin = tex1Dfetch(gf_tex,2*pos + 1);
  
  // reconstruct a1
  help = sqrtf(1.0f - help);
  (*gf)[0][0].re = help*cosf(gfin.x);
  (*gf)[0][0].im = help*sinf(gfin.x);
  
  // assign b1
  (*gf)[1][0].re = gfin.z;
  (*gf)[1][0].im = gfin.w;
  
  // p2 = 1/N b1
  p2.re = one_over_N*(*gf)[1][0].re;
  p2.im = one_over_N*(*gf)[1][0].im;  


  // reconstruct c1
  help = sqrtf(1.0 - 
              (*gf)[0][0].re * (*gf)[0][0].re - (*gf)[0][0].im * (*gf)[0][0].im - 
              (*gf)[1][0].re * (*gf)[1][0].re - (*gf)[1][0].im * (*gf)[1][0].im
          );
  (*gf)[2][0].re = help*cosf(gfin.y);
  (*gf)[2][0].im = help*sinf(gfin.y);

  
  // p1 = 1/N*cconj(c1)
  p1.re = one_over_N*(*gf)[2][0].re;
  p1.im = - one_over_N*(*gf)[2][0].im;
  
  
  // calculate b2
  chelp1 = dev_cmult(p1,  dev_cconj( (*gf)[0][2] )   );
  chelp2 = dev_cmult(p2, dev_cmult( dev_cconj((*gf)[0][0]) , (*gf)[0][1] )  );
  chelp1 = dev_cadd(chelp1, chelp2);
  (*gf)[1][1] = dev_crealmult(chelp1, -one_over_N);
  
  // calculate b3
  chelp1 = dev_cmult(p1,  dev_cconj( (*gf)[0][1] )   );
  chelp2 = dev_cmult(p2, dev_cmult( dev_cconj((*gf)[0][0]) , (*gf)[0][2] )  );
  chelp1 = dev_cadd(chelp1, chelp2);
  (*gf)[1][2] = dev_crealmult(chelp1, -one_over_N);
  
  // calculate c2
  chelp1 = dev_cmult(  dev_cconj(p2) ,  dev_cconj( (*gf)[0][2] )   );
  chelp2 = dev_cmult(  dev_cconj(p1) , 
                       dev_cmult(  dev_cconj( (*gf)[0][0] ) , (*gf)[0][1] )
                     );
  chelp1 = dev_csub(chelp1, chelp2);
  (*gf)[2][1] = dev_crealmult(chelp1, one_over_N);
  
  
  // calculate c3
  chelp1 = dev_cmult(  dev_cconj(p2) ,  dev_cconj( (*gf)[0][1] )   );
  chelp2 = dev_cmult(  dev_cconj(p1) , 
                       dev_cmult(  dev_cconj( (*gf)[0][0] ) , (*gf)[0][2] )
                     );
  chelp1 = dev_csub(chelp1, chelp2);
  (*gf)[2][2] = dev_crealmult(chelp1, one_over_N);




  // till here this was the reconstruction of the link (not daggered)
  // calculate now U^dagger trivially
  // this should be implemented more efficiently !!
  
  
  //diagonal
  (*gf)[0][0] = dev_cconj((*gf)[0][0]);
  (*gf)[1][1] = dev_cconj((*gf)[1][1]);
  (*gf)[2][2] = dev_cconj((*gf)[2][2]);
  
  //off-diagonals
  chelp1 = dev_cconj(  (*gf)[0][1] );
  (*gf)[0][1] = dev_cconj ( (*gf)[1][0]  );
  (*gf)[1][0] = chelp1;
  
  chelp1 = dev_cconj(  (*gf)[0][2] );
  (*gf)[0][2] = dev_cconj ( (*gf)[2][0]  );
  (*gf)[2][0] = chelp1;
  
  chelp1 = dev_cconj(  (*gf)[1][0] );
  (*gf)[1][0] = dev_cconj ( (*gf)[0][1]  );
  (*gf)[0][1] = chelp1;
  
}







__global__ void dev_gamma5(dev_spinor * sin, dev_spinor * sout){
  int pos;
  pos= threadIdx.x + blockDim.x*blockIdx.x;  
  if(pos < dev_VOLUME){
          sout[6*pos+0].x = sin[6*pos+0].x;
          sout[6*pos+0].y = sin[6*pos+0].y;
          sout[6*pos+0].z = sin[6*pos+0].z;
          sout[6*pos+0].w = sin[6*pos+0].w;
          sout[6*pos+1].x = sin[6*pos+1].x;
          sout[6*pos+1].y = sin[6*pos+1].y;
          
          sout[6*pos+1].z = sin[6*pos+1].z;
          sout[6*pos+1].w = sin[6*pos+1].w;
          sout[6*pos+2].x = sin[6*pos+2].x;
          sout[6*pos+2].y = sin[6*pos+2].y;
          sout[6*pos+2].z = sin[6*pos+2].z;
          sout[6*pos+2].w = sin[6*pos+2].w;   
          
          sout[6*pos+3].x = -1.0*sin[6*pos+3].x;
          sout[6*pos+3].y = -1.0*sin[6*pos+3].y;
          sout[6*pos+3].z = -1.0*sin[6*pos+3].z;
          sout[6*pos+3].w = -1.0*sin[6*pos+3].w;
          sout[6*pos+4].x = -1.0*sin[6*pos+4].x;
          sout[6*pos+4].y = -1.0*sin[6*pos+4].y; 

          sout[6*pos+4].z = -1.0*sin[6*pos+4].z;
          sout[6*pos+4].w = -1.0*sin[6*pos+4].w;
          sout[6*pos+5].x = -1.0*sin[6*pos+5].x;
          sout[6*pos+5].y = -1.0*sin[6*pos+5].y;
          sout[6*pos+5].z = -1.0*sin[6*pos+5].z;
          sout[6*pos+5].w = -1.0*sin[6*pos+5].w;                 
  } 
}




__global__ void dev_swapmu(){
  if(blockIdx.x == 0 && threadIdx.x == 0){
    mu = - mu;
  }
}


// computes sout = 1/(1 +- mutilde gamma5) sin = (1 -+ i mutilde gamma5)/(1+mutilde^2) sin
// mutilde = 2 kappa mu
__global__ void dev_mul_one_pm_imu_inv(dev_spinor* sin, dev_spinor* sout, const REAL sign){
  
   //need the inverse sign in the numerator because of inverse
   dev_complex pm_imu = dev_initcomplex(0.0,-1.0*sign*twokappamu);
   
   REAL one_plus_musquare_inv = 1.0/(1.0 + twokappamu*twokappamu);
   int pos;
   pos= threadIdx.x + blockDim.x*blockIdx.x;  
   if(pos < dev_VOLUME){
     dev_skalarmult_spinor(&(sin[6*pos]), pm_imu, &(sout[6*pos]));
     dev_Gamma5(&(sout[6*pos]));
     dev_add_spinor_assign(&(sout[6*pos]), &(sin[6*pos]));
     dev_realmult_spinor(&(sout[6*pos]), one_plus_musquare_inv);
   }
}

// sout = gamma_5*((1\pm i\mutilde \gamma_5)*sin1 - sin2)
__global__ void dev_mul_one_pm_imu_sub_mul_gamma5(dev_spinor* sin1, dev_spinor* sin2, dev_spinor* sout, const REAL sign){

   dev_complex pm_imu = dev_initcomplex(0.0, sign*twokappamu); // i mutilde
   int pos;
   pos= threadIdx.x + blockDim.x*blockIdx.x;  
   if(pos < dev_VOLUME){
     dev_skalarmult_spinor(&(sin1[6*pos]), pm_imu, &(sout[6*pos]));
     dev_Gamma5(&(sout[6*pos]));
     dev_add_spinor_assign(&(sout[6*pos]), &(sin1[6*pos]));
     dev_sub_spinor_assign(&(sout[6*pos]), &(sin2[6*pos]));
     dev_Gamma5(&(sout[6*pos]));
   }   
}




//applies the Hopping Part Even-Odd !
//the gauge field is the complete gaugefield!
//the gauge field at the local point is reconstructed by 2*pos+eo where pos is the eo-position
//from 0..VOLUME/2-1, eo = 0 or 1
//the positions in the gauge fields are passed in "gfindex_site" for gf's that are attached at
//the actual positions and in "gfindex_nextsite" for gf's that start at a position of the 
//other eo-sublattice.
//for the hopping positions of the eo-spinor field we use on of the two dedicated eo-nn fields
//the boundary conditions are implemented as in Hopping_Matrix.c
//mult with complex conjugate k0,k1,k2,k3 in positive direction because
// psi(x+mu) != exp(i theta_mu) psi(x)  
__global__ void dev_Hopping_Matrix(dev_su3_2v * gf, dev_spinor * sin, dev_spinor * sout, int * gfindex_site,int* gfindex_nextsite, int * nn_evenodd, const int eo){

  int pos,hoppos;
    dev_spinor shelp1[6], ssum[6];
    __shared__ dev_su3 gfsmem[BLOCK];
    

  pos= threadIdx.x + blockDim.x*blockIdx.x;  
  int ix = threadIdx.x;
  if(pos < dev_VOLUME){

  dev_zero_spinor(&(ssum[0])); // zero sum        
//hopping term                
//l==0,t
            //positive direction
            hoppos = nn_evenodd[8*pos];
            //color
            dev_reconstructgf_2vtexref(4*(gfindex_site[pos]),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r - gamma_mu)
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk0,&(shelp1[0]), &(ssum[0]));
            dev_Gamma0(&(shelp1[0]));
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_k0,&(shelp1[0]), &(ssum[0]));

            //negative direction
            hoppos = nn_evenodd[8*pos+4]; 
            //color
            dev_reconstructgf_2vtexref_dagger(4*gfindex_nextsite[hoppos],&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));     
            //-kappa(r + gamma_mu)
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk0,&(shelp1[0]), &(ssum[0]));
            dev_Gamma0(&(shelp1[0]));
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk0,&(shelp1[0]), &(ssum[0]));


//l==3,z               
            //positive direction
            hoppos = nn_evenodd[8*pos+3];
            //color
            dev_reconstructgf_2vtexref(4*(gfindex_site[pos])+(3),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r - gamma_mu)
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk3,&(shelp1[0]), &(ssum[0]));
            dev_Gamma3(&(shelp1[0]));
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_k3,&(shelp1[0]), &(ssum[0]));

            //negative direction
            hoppos = nn_evenodd[8*pos+7]; 
            //color
            dev_reconstructgf_2vtexref_dagger(4*gfindex_nextsite[hoppos]+(3),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r + gamma_mu)
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk3,&(shelp1[0]), &(ssum[0]));
            dev_Gamma3(&(shelp1[0]));
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk3,&(shelp1[0]), &(ssum[0]));
         
         
//l==2,y        
            //positive direction
            hoppos = nn_evenodd[8*pos+2];
            //color
            dev_reconstructgf_2vtexref(4*(gfindex_site[pos])+(2),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r - gamma_mu)
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk2,&(shelp1[0]), &(ssum[0]));
            dev_Gamma2(&(shelp1[0]));
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_k2,&(shelp1[0]), &(ssum[0]));
            
            //negative direction
            hoppos = nn_evenodd[8*pos+6]; 
            //color
            dev_reconstructgf_2vtexref_dagger(4*gfindex_nextsite[hoppos]+(2),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r + gamma_mu)
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk2,&(shelp1[0]), &(ssum[0]));
            dev_Gamma2(&(shelp1[0]));
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk2,&(shelp1[0]), &(ssum[0]));


//l==1,x 
            //positive direction
            hoppos = nn_evenodd[8*pos+1];
            //color
            dev_reconstructgf_2vtexref(4*(gfindex_site[pos])+(1),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r - gamma_mu)
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk1,&(shelp1[0]), &(ssum[0]));
            dev_Gamma1(&(shelp1[0]));
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_k1,&(shelp1[0]), &(ssum[0]));

            //negative direction
            hoppos = nn_evenodd[8*pos+5]; 
            //color
            dev_reconstructgf_2vtexref_dagger(4*gfindex_nextsite[hoppos]+(1),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r + gamma_mu)
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk1,&(shelp1[0]), &(ssum[0]));
            dev_Gamma1(&(shelp1[0]));
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk1,&(shelp1[0]), &(ssum[0]));      
               
 
        //copy to output spinor
        dev_copy_spinor(&(ssum[0]),&(sout[6*pos])); 
  }
}





// aequivalent to Qtm_pm_psi in tm_operators.c
extern "C" void dev_Qtm_pm_psi(dev_spinor* spinin, dev_spinor* spinout, int gridsize, int blocksize){
  //spinin == odd
  //spinout == odd
  
  //Q_{-}
  bind_texture_spin(spinin,1);
    dev_Hopping_Matrix<<<gridsize, blocksize>>>
             (dev_gf, spinin, dev_spin_eo1, dev_eoidx_even, dev_eoidx_odd, dev_nn_eo, 0); //dev_spin_eo1 == even -> 0
  unbind_texture_spin(1);
  dev_mul_one_pm_imu_inv<<<gridsize, blocksize>>>(dev_spin_eo1,dev_spin_eo2, -1.);
  
  bind_texture_spin(dev_spin_eo2,1);
    dev_Hopping_Matrix<<<gridsize, blocksize>>>
            (dev_gf, dev_spin_eo2, dev_spin_eo1, dev_eoidx_odd, dev_eoidx_even, dev_nn_oe, 1); 
  unbind_texture_spin(1);
  dev_mul_one_pm_imu_sub_mul_gamma5<<<gridsize, blocksize>>>(spinin, dev_spin_eo1,  dev_spin_eo2, -1.);
  
  //Q_{+}
  bind_texture_spin(dev_spin_eo2,1);
    dev_Hopping_Matrix<<<gridsize, blocksize>>>
          (dev_gf, dev_spin_eo2, dev_spin_eo1, dev_eoidx_even, dev_eoidx_odd, dev_nn_eo, 0); //dev_spin_eo1 == even -> 0
  unbind_texture_spin(1);
  dev_mul_one_pm_imu_inv<<<gridsize, blocksize>>>(dev_spin_eo1,spinout, +1.);
  
  bind_texture_spin(spinout,1);
    dev_Hopping_Matrix<<<gridsize, blocksize>>>
             (dev_gf, spinout, dev_spin_eo1, dev_eoidx_odd, dev_eoidx_even, dev_nn_oe, 1); 
  unbind_texture_spin(1);
  dev_mul_one_pm_imu_sub_mul_gamma5<<<gridsize, blocksize>>>(dev_spin_eo2, dev_spin_eo1,  spinout , +1.); 
  
}






//applies the full tm Operator
// uses texture cache (spin_tex) for input spinor
// runs through whole lattice for output spinor
// D_psi uses phase_mu and not ka_mu for the boundary conds (vice versa in HoppingMatrix) 
// -> thats why complexmult and complexcgmult are interchanged in dev_HoppingMatrix and in 
// dev_tm_dirac_kappa
__global__ void dev_tm_dirac_kappa(dev_su3_2v * gf, dev_spinor * sin, dev_spinor * sout, int * dev_nn){
    int pos,hoppos;
    dev_spinor shelp1[6], ssum[6];
    __shared__ dev_su3 gfsmem[BLOCK];
    

  pos= threadIdx.x + blockDim.x*blockIdx.x;  
  int ix = threadIdx.x;
  if(pos < dev_VOLUME){
        
          //dev_zero_spinor(&(ssum[0])); // zero sum
          //skalarer Term
          ssum[0] = tex1Dfetch(spin_tex,6*pos);
          ssum[1] = tex1Dfetch(spin_tex,6*pos+1);
          ssum[2] = tex1Dfetch(spin_tex,6*pos+2);
          ssum[3] = tex1Dfetch(spin_tex,6*pos+3);
          ssum[4] = tex1Dfetch(spin_tex,6*pos+4);
          ssum[5] = tex1Dfetch(spin_tex,6*pos+5);

          
//hopping term                
//l==0,t
            //positive direction
            hoppos = dev_nn[8*pos];
            //color
            dev_reconstructgf_2vtexref(4*pos,&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r - gamma_mu)
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk0,&(shelp1[0]), &(ssum[0]));
            //dev_GammatV(0,&(shelp1[0]));
            dev_Gamma0(&(shelp1[0]));
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_k0,&(shelp1[0]), &(ssum[0]));

            //negative direction
            hoppos = dev_nn[8*pos+4];
            //color
            dev_reconstructgf_2vtexref_dagger(4*hoppos,&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));     
            //-kappa(r + gamma_mu)
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk0,&(shelp1[0]), &(ssum[0]));
            //dev_GammatV(0,&(shelp1[0]));
            dev_Gamma0(&(shelp1[0]));
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk0,&(shelp1[0]), &(ssum[0]));


//l==3,z               
            //positive direction
            hoppos = dev_nn[8*pos+3];
            //color
            dev_reconstructgf_2vtexref(4*pos+(3),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r - gamma_mu)
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk3,&(shelp1[0]), &(ssum[0]));
            //dev_GammatV(3,&(shelp1[0]));
            dev_Gamma3(&(shelp1[0]));
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_k3,&(shelp1[0]), &(ssum[0]));

            //negative direction
            hoppos = dev_nn[8*pos+7];
            //color
            dev_reconstructgf_2vtexref_dagger(4*hoppos+(3),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r + gamma_mu)
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk3,&(shelp1[0]), &(ssum[0]));
            //dev_GammatV(3,&(shelp1[0]));
            dev_Gamma3(&(shelp1[0]));
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk3,&(shelp1[0]), &(ssum[0]));
         
         
//l==2,y        
            //positive direction
            hoppos = dev_nn[8*pos+2];
            //color
            dev_reconstructgf_2vtexref(4*pos+(2),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r - gamma_mu)
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk2,&(shelp1[0]), &(ssum[0]));
            //dev_GammatV(2,&(shelp1[0]));
            dev_Gamma2(&(shelp1[0]));
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_k2,&(shelp1[0]), &(ssum[0]));
            
            //negative direction
            hoppos = dev_nn[8*pos+6];
            //color
            dev_reconstructgf_2vtexref_dagger(4*hoppos+(2),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r + gamma_mu)
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk2,&(shelp1[0]), &(ssum[0]));
            //dev_GammatV(2,&(shelp1[0]));
            dev_Gamma2(&(shelp1[0]));
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk2,&(shelp1[0]), &(ssum[0]));


//l==1,x 
            //positive direction
            hoppos = dev_nn[8*pos+1];
            //color
            dev_reconstructgf_2vtexref(4*pos+(1),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r - gamma_mu)
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_mk1,&(shelp1[0]), &(ssum[0]));
            //dev_GammatV(1,&(shelp1[0]));
            dev_Gamma1(&(shelp1[0]));
            dev_complexmult_add_assign_spinor(&(ssum[0]),dev_k1,&(shelp1[0]), &(ssum[0]));

            //negative direction
            hoppos = dev_nn[8*pos+5];
            //color
            dev_reconstructgf_2vtexref_dagger(4*hoppos+(1),&(gfsmem[ix]));
            dev_su3MtV_spintex(gfsmem[ix], hoppos, &(shelp1[0]));
            //-kappa(r + gamma_mu)
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk1,&(shelp1[0]), &(ssum[0]));
            //dev_GammatV(1,&(shelp1[0]));
            dev_Gamma1(&(shelp1[0]));
            dev_complexcgmult_add_assign_spinor(&(ssum[0]),dev_mk1,&(shelp1[0]), &(ssum[0]));  
          
          
          
          //gamma5 term
          shelp1[0] = tex1Dfetch(spin_tex,6*pos);
          shelp1[1] = tex1Dfetch(spin_tex,6*pos+1);
          shelp1[2] = tex1Dfetch(spin_tex,6*pos+2);
          shelp1[3] = tex1Dfetch(spin_tex,6*pos+3);
          shelp1[4] = tex1Dfetch(spin_tex,6*pos+4);
          shelp1[5] = tex1Dfetch(spin_tex,6*pos+5);
          
          
          
          //dev_GammatV(4,&(shelp1[0]));
          dev_Gamma5(&(shelp1[0]));
          dev_complexmult_add_assign_spinor(&(ssum[0]),dev_initcomplex(0.0,2.0*kappa*mu),&(shelp1[0]), &(sout[6*pos]));
  }
}





extern "C" void dev_tm_dirac_dagger_kappa(dev_su3_2v * gf,dev_spinor* spinin, dev_spinor* spinout, 
 int *grid, int * nn_grid, REAL* output,REAL* erg, int xsize, int ysize){
 int gridsize;
 if( VOLUME >= 128){
   gridsize =VOLUME/128;
 }
 else{
   gridsize=1;
 }
 dim3 griddim2(gridsize,1,1);
 dim3 blockdim2(128,1,1);
 dim3 blockdim(xsize,ysize);
 
  dim3 blockdim3(BLOCK,1,1);
 if( VOLUME >= BLOCK){
   gridsize =VOLUME/BLOCK;
 }
 else{
   gridsize=1;
 }
 dim3 griddim3(gridsize,1,1); 
  dev_gamma5 <<<griddim2, blockdim2 >>> (spinin,spinout);
  dev_tm_dirac_kappa <<<griddim3, blockdim3 >>> (gf, spinout, spinin, dev_nn);
  dev_gamma5 <<<griddim2, blockdim2 >>>(spinin,spinout);
}







__device__ inline REAL dev_skalarprod_spinor(dev_spinor * s1, dev_spinor * s2){
  REAL skalprod = 0.0;
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    skalprod += ((*(s1+i)).x*(*(s2+i)).x + (*(s1+i)).y*(*(s2+i)).y + (*(s1+i)).z*(*(s2+i)).z + (*(s1+i)).w*(*(s2+i)).w);
  }
  return skalprod;
}




__device__ inline REAL dev_squarenorm_spinor(dev_spinor * s1){
  REAL skalprod = 0.0;
  int i;
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    skalprod += ((*(s1+i)).x*(*(s1+i)).x + (*(s1+i)).y*(*(s1+i)).y + (*(s1+i)).z*(*(s1+i)).z + (*(s1+i)).w*(*(s1+i)).w);
  }
  return skalprod;
}



__device__ inline REAL dev_squarenorm_spinor_tex(int pos){
  REAL skalprod = 0.0;
  int i;
  float4 help;
  
  #pragma unroll 6
  for(i=0;i<6;i++){ //color + spin
    help = tex1Dfetch(spin_tex2,6*pos+i);
    skalprod += help.x*help.x + help.y*help.y + help.z*help.z + help.w*help.w;
  }
  return skalprod;
}




//only 1 dim parallel possible, because need __syncthread !
__global__ void dev_skalarprod_spinor_field2(dev_spinor* s1, dev_spinor* s2, REAL* erg){
  __shared__ REAL shrinkarray[ACCUM_N];
  int pos,stepwidth;
  REAL ks,kc,ds,tr,ts,tt;
  
   
   // ADD ERROR HERE if t > maxblockdim
   
   ks=0.0;
   kc=0.0; 
   
   if(blockDim.x > dev_VOLUME){
     stepwidth = 1;  
   }
   else{
     stepwidth = dev_VOLUME/(gridDim.x*blockDim.x);
   }
   
     int start = (blockIdx.x*blockDim.x + threadIdx.x)*stepwidth;
     int end = (blockIdx.x*blockDim.x + threadIdx.x+1)*stepwidth;  
   
   for(pos=start;pos<end; pos++){
     if(pos < dev_VOLUME){
          ds = dev_skalarprod_spinor(&(s1[6*pos]), &(s2[6*pos] ));
      }
      else{
        ds = 0.0;
      } 
          // Kahan summation
          tr=ds+kc;
          ts=tr+ks;
          tt=ts-ks;
          ks=ts;
          kc=tr-tt;
    }
    kc=ks+kc;
    shrinkarray[threadIdx.x] = kc;
    __syncthreads();
    
    
    if(threadIdx.x==0){
      ks=0.0;
      kc=0.0; 
      int k;
      for(k=0; k<blockDim.x; k++){
        ds = shrinkarray[k];
        tr=ds+kc;
        ts=tr+ks;
        tt=ts-ks;
        ks=ts;
        kc=tr-tt;
      }
      kc=ks+kc;
      (erg[blockIdx.x])=kc;
    }//threadIdx==0;
}








//only 1 dim parallel possible, because need __syncthread !
__global__ void dev_squarenorm_spinor_field(dev_spinor* s1, REAL* erg){
  __shared__ REAL shrinkarray[ACCUM_N];
  int pos,stepwidth;
  REAL ks,kc,ds,tr,ts,tt;
  
   
   // ADD ERROR HERE if t > maxblockdim
   
   ks=0.0;
   kc=0.0; 
   
   if(blockDim.x > dev_VOLUME){
     stepwidth = 1;  
   }
   else{
     stepwidth = dev_VOLUME/(gridDim.x*blockDim.x);
   }
   
     int start = (blockIdx.x*blockDim.x + threadIdx.x)*stepwidth;
     int end = (blockIdx.x*blockDim.x + threadIdx.x+1)*stepwidth;  
   
   for(pos=start;pos<end; pos++){
     if(pos < dev_VOLUME){
         //ds = dev_squarenorm_spinor_tex(pos);
         ds = dev_squarenorm_spinor(&(s1[6*pos]));
      }
      else{
        ds = 0.0;
      } 
          // Kahan summation
          tr=ds+kc;
          ts=tr+ks;
          tt=ts-ks;
          ks=ts;
          kc=tr-tt;
    }
    kc=ks+kc;
    shrinkarray[threadIdx.x] = kc;
    __syncthreads();
    
    
    if(threadIdx.x==0){
      ks=0.0;
      kc=0.0; 
      int k;
      for(k=0; k<blockDim.x; k++){
        ds = shrinkarray[k];
        tr=ds+kc;
        ts=tr+ks;
        tt=ts-ks;
        ks=ts;
        kc=tr-tt;
      }
      kc=ks+kc;
      (erg[blockIdx.x])=kc;
    }//threadIdx==0;
}






//only 1 dim parallel, because need __syncthread !
__global__ void dev_skalarprod_spinor_field(dev_spinor* s1, dev_spinor* s2, REAL* erg){
  __shared__ REAL shrinkarray[ACCUM_N];
  int pos,stepwidth, sweepsperthread;
  REAL ks,kc,ds,tr,ts,tt;
   
   // ADD ERROR HERE if t > maxblockdim
   
   ks=0.0;
   kc=0.0; 
   
   if(ACCUM_N > dev_VOLUME){
     stepwidth = 1;
     sweepsperthread = 1;  
   }
   else{
     stepwidth = dev_VOLUME/ACCUM_N;
     sweepsperthread = ACCUM_N/blockDim.x;
   }
    
   
   
 for(int j = 0; j < sweepsperthread; j++){
   
     int start = (threadIdx.x + j*blockDim.x)*stepwidth;
     int end = (threadIdx.x+j*blockDim.x+1)*stepwidth;  
     ks=0.0;
     kc=0.0; 
     
   for(pos=start;pos<end; pos++){
     if(pos < dev_VOLUME){
          ds = dev_skalarprod_spinor(&(s1[6*pos]), &(s2[6*pos] ));
          
      }
      else{
        ds = 0.0;
      } 
          // Kahan summation
          tr=ds+kc;
          ts=tr+ks;
          tt=ts-ks;
          ks=ts;
          kc=tr-tt;
    }
    kc=ks+kc;
    shrinkarray[threadIdx.x+j*blockDim.x] = kc;
  }
  __syncthreads();
   
 
    for(int stride = ACCUM_N / 2; stride > 0; stride >>= 1){
       __syncthreads();
       for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x)
           shrinkarray[iAccum] += shrinkarray[stride + iAccum];
    }

    if(threadIdx.x == 0) (*erg) = shrinkarray[0];
    
    
    /*
    if(threadIdx.x==0){
      ks=0.0;
      kc=0.0; 
      int k;
      for(k=0; k<sweepsperthread*blockDim.x; k++){
        ds = shrinkarray[k];
        tr=ds+kc;
        ts=tr+ks;
        tt=ts-ks;
        ks=ts;
        kc=tr-tt;
      }
      kc=ks+kc;
      (*erg)=kc;
    }//threadIdx==0;
    */
    
}




__global__ void dev_zero_spinor_field(dev_spinor* s1){
  int pos;
  pos= threadIdx.x + blockDim.x*blockIdx.x;  
  if(pos < dev_VOLUME){
          dev_zero_spinor(&(s1[6*pos]));
  }
}




__global__ void dev_copy_spinor_field(dev_spinor* s1, dev_spinor* s2){
    int pos;
  pos= threadIdx.x + blockDim.x*blockIdx.x;  
  if(pos < dev_VOLUME){
      dev_copy_spinor(&(s1[6*pos]),&(s2[6*pos]));
  } 
}



__global__ void dev_skalarmult_add_assign_spinor_field(dev_spinor* s1, REAL lambda, dev_spinor* s2, dev_spinor* so){
  int pos;
  pos= threadIdx.x + blockDim.x*blockIdx.x;  
  if(pos < dev_VOLUME){
    dev_skalarmult_add_assign_spinor(&(s1[6*pos]), lambda ,&(s2[6*pos]), &(so[6*pos]) );
  }
}



__global__ void dev_skalarmult_spinor_field(dev_spinor* s1, REAL lambda, dev_spinor* so){
  int pos;
  pos= threadIdx.x + blockDim.x*blockIdx.x;  
  if(pos < dev_VOLUME){
    dev_skalarmult_spinor(&(s1[6*pos]), dev_initcomplex(lambda,0.0) , &(so[6*pos]) );
  }
}  



__global__ void dev_complexmult_spinor_field(dev_spinor* s1, dev_complex lambda, dev_spinor* so){
  int pos;
  pos= threadIdx.x + blockDim.x*blockIdx.x;  
  if(pos < dev_VOLUME){
    dev_skalarmult_spinor(&(s1[6*pos]), lambda , &(so[6*pos]) );
  }
}






// init the gpu inner solver, assigen constants etc.
__global__ void he_cg_init (int* grid, REAL param_kappa, REAL param_mu, dev_complex k0, dev_complex k1, dev_complex k2, dev_complex k3){
  dev_LX = grid[0];
  dev_LY = grid[1];
  dev_LZ = grid[2];
  dev_T = grid[3];
  dev_VOLUME = grid[4]; // grid[4] is initialized 1/2 VOLUME for eo
  
  kappa = param_kappa;
  mu = param_mu;
  twokappamu = 2.0*param_kappa*param_mu;
  
  dev_k0.re = k0.re;
  dev_k0.im = k0.im;
  dev_mk0.re = -k0.re;
  dev_mk0.im = -k0.im;
  
  dev_k1.re = k1.re;
  dev_k1.im = k1.im;
  dev_mk1.re = -k1.re;
  dev_mk1.im = -k1.im;
  
  dev_k2.re = k2.re;
  dev_k2.im = k2.im;
  dev_mk2.re = -k2.re;
  dev_mk2.im = -k2.im;
  
  dev_k3.re = k3.re;
  dev_k3.im = k3.im;
  dev_mk3.re = -k3.re;
  dev_mk3.im = -k3.im;
}




// code to list available devices, not yet included in main code
// this is copied from the CUDA sdk 
extern "C" int find_devices(){
int deviceCount, dev;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999)
                printf("There is no device supporting CUDA.\n");
            else if (deviceCount == 1)
                printf("There is 1 device supporting CUDA\n");
            else
                printf("There are %d devices supporting CUDA\n", deviceCount);
        }
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  Major revision number:                         %d\n",
               deviceProp.major);
        printf("  Minor revision number:                         %d\n",
               deviceProp.minor);
        printf("  Total amount of global memory:                 %u bytes\n",
               deviceProp.totalGlobalMem);
    #if CUDART_VERSION >= 2000
        printf("  Number of multiprocessors:                     %d\n",
               deviceProp.multiProcessorCount);
        printf("  Number of cores:                               %d\n",
               8 * deviceProp.multiProcessorCount);
    #endif
        printf("  Total amount of constant memory:               %u bytes\n",
               deviceProp.totalConstMem); 
        printf("  Total amount of shared memory per block:       %u bytes\n",
               deviceProp.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n",
               deviceProp.regsPerBlock);
        printf("  Warp size:                                     %d\n",
               deviceProp.warpSize);
        printf("  Maximum number of threads per block:           %d\n",
               deviceProp.maxThreadsPerBlock);
        printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("  Maximum memory pitch:                          %u bytes\n",
               deviceProp.memPitch);
        printf("  Texture alignment:                             %u bytes\n",
               deviceProp.textureAlignment);
        printf("  Clock rate:                                    %.2f GHz\n",
               deviceProp.clockRate * 1e-6f);
    #if CUDART_VERSION >= 2000
        printf("  Concurrent copy and execution:                 %s\n",
               deviceProp.deviceOverlap ? "Yes" : "No");
    #endif
    }
    return(deviceCount);
}






extern "C" int bind_texture_spin(dev_spinor* s, int i){
  
  size_t size;
  if(even_odd_flag){
    size = sizeof(float4)*6*VOLUME/2;
  }
  else{
    size = sizeof(float4)*6*VOLUME;
  }
   
  
  switch(i){
    case 1:
      //printf("Binding texture to spinorfield 1\n");
      spin_texRefPtr = NULL;
      hipGetTextureReference(&spin_texRefPtr, HIP_SYMBOL("spin_tex"));
      spin_channelDesc =  hipCreateChannelDesc<float4>();
      hipBindTexture(0, spin_texRefPtr, s, &spin_channelDesc, size);
      //printf("%s\n", hipGetErrorString(hipGetLastError())); 
      return(0);
   
    case 2:
      //printf("Binding texture to spinorfield 2\n");
      spin_texRefPtr2 = NULL;
      hipGetTextureReference(&spin_texRefPtr2, HIP_SYMBOL("spin_tex2"));
      spin_channelDesc2 =  hipCreateChannelDesc<float4>();
      hipBindTexture(0, spin_texRefPtr2, s, &spin_channelDesc2, size);
      //printf("%s\n", hipGetErrorString(hipGetLastError()));  
      return(0);
  }
return(1);  
}


extern "C" int unbind_texture_spin(int i){
  switch(i){
    case 1:
      //printf("Unbinding texture of spinorfield 1\n");
      hipUnbindTexture(spin_texRefPtr);
      //printf("%s\n", hipGetErrorString(hipGetLastError()));    
      return(0);
    case 2:
      //printf("Unbinding texture of spinorfield 2\n");
      hipUnbindTexture(spin_texRefPtr2);
      //printf("%s\n", hipGetErrorString(hipGetLastError()));    
      return(0);    
  }
  
return(1);
}



extern "C" int bind_texture_gf(dev_su3_2v * gf){
 //printf("Binding texture to gaugefield\n");
 size_t size = sizeof(float4)*3*VOLUME*4;
 hipGetTextureReference(&gf_texRefPtr, HIP_SYMBOL("gf_tex"));
 gf_channelDesc =  hipCreateChannelDesc<float4>();
 hipBindTexture(0, gf_texRefPtr, gf, &gf_channelDesc, size);
 printf("%s\n", hipGetErrorString(hipGetLastError()));    
 return(0);
}


extern "C" int unbind_texture_gf(){
 //printf("Unbinding texture to gaugefield\n");
 hipUnbindTexture(gf_texRefPtr);
 printf("%s\n", hipGetErrorString(hipGetLastError()));    
 return(0);
}







extern "C" void test_operator(dev_su3_2v * gf,dev_spinor* spinin, dev_spinor* spinout, 
dev_spinor* spin0, dev_spinor* spin1, dev_spinor* spin2, dev_spinor* spin3, dev_spinor* spin4, int *grid, int * nn_grid, REAL* output,REAL* erg, int xsize, int ysize){
 
 int  gridsize;

 dim3 blockdim(1,1);
 dim3 blockdim2(128,1,1);
 if( VOLUME >= 128){
   gridsize =VOLUME/128;
 }
 else{
   gridsize=1;
 }
 dim3 griddim2(gridsize,1,1);
 
 
 dim3 blockdim3(BLOCK,1,1);
 if( VOLUME >= BLOCK){
   gridsize =VOLUME/BLOCK;
 }
 else{
   gridsize=1;
 }
 dim3 griddim3(gridsize,1,1); 
 
 
  dev_complex h0,h1,h2,h3;
  h0.re = (REAL)ka0.re;    h0.im = (REAL)ka0.im;
  h1.re = (REAL)ka1.re;    h1.im = (REAL)ka1.im;
  h2.re = (REAL)ka2.re;    h2.im = (REAL)ka2.im;
  h3.re = (REAL)ka3.re;    h3.im = (REAL)ka3.im;
  he_cg_init<<< 1, 1 >>> (grid, (REAL) g_kappa, (REAL)(g_mu/(2.0*g_kappa)), h0,h1,h2,h3);
 
 
  REAL scaleparam = sqrt(1.0/(2.0 * (REAL) hostkappa));
  dev_skalarmult_spinor_field<<<griddim2, blockdim2 >>>(spinin,scaleparam*scaleparam, spin4);
  
 bind_texture_gf(gf);
 bind_texture_spin(spin4,1);
  
  // apply D_tm
  dev_tm_dirac_kappa <<<griddim3, blockdim3 >>>(gf, spin4, spinout, nn_grid);


  unbind_texture_gf();
  unbind_texture_spin(1);
}






extern "C" void dev_cg(dev_su3_2v * gf,dev_spinor* spinin, dev_spinor* spinout, 
dev_spinor* spin0, dev_spinor* spin1, dev_spinor* spin2, dev_spinor* spin3, dev_spinor* spin4, int *grid, int * nn_grid, REAL* output,REAL* erg, int xsize, int ysize, int rescalekappa){
 
 
 REAL host_alpha, host_beta, host_dotprod, host_rk, sourcesquarenorm;
 REAL * dotprod, * dotprod2, * rk, * alpha, *beta;
 
 
 
 int i, gridsize;
 int maxit = max_innersolver_it;
 REAL eps = (REAL) innersolver_precision;
 int N_recalcres = 10; // after N_recalcres iterations calculate r = A x_k - b
 
 
 // initialize grid and block, make sure VOLUME is a multiple of blocksize 
 if(VOLUME%DOTPROD_DIM != 0){
   printf("Error: VOLUME is not a multiple of DOTPROD_DIM. Aborting...\n");
   exit(100); 
 }

 dim3 blockdim(1,1);
 dim3 blockdim2(128,1,1);
 if( VOLUME >= 128){
   gridsize =VOLUME/128;
 }
 else{
   gridsize=1;
 }
 dim3 griddim2(gridsize,1,1);
 
 
 if(VOLUME%BLOCK != 0){
   printf("Error: VOLUME is not a multiple of BLOCK. Aborting...\n");
   exit(100);
 }
 dim3 blockdim3(BLOCK,1,1);
 if( VOLUME >= BLOCK){
   gridsize =VOLUME/BLOCK;
 }
 else{
   gridsize=1;
 }
 dim3 griddim3(gridsize,1,1); 
 
 size_t size2 = sizeof(float4)*6*VOLUME;
 
 //Bind texture gf
 bind_texture_gf(gf);
 //Bind texture spinor to spin4 (D_tm is always applied to spin4)
 bind_texture_spin(spin4,1);
 
 
 //Initialize some stuff
  printf("mu = %f\n", g_mu);
  dev_complex h0,h1,h2,h3;
  h0.re = (REAL)ka0.re;    h0.im = (REAL)ka0.im;
  h1.re = (REAL)ka1.re;    h1.im = (REAL)ka1.im;
  h2.re = (REAL)ka2.re;    h2.im = (REAL)ka2.im;
  h3.re = (REAL)ka3.re;    h3.im = (REAL)ka3.im;
  he_cg_init<<< 1, 1 >>> (grid, (REAL) g_kappa, (REAL)(g_mu/(2.0*g_kappa)), h0,h1,h2,h3);
  // BEWARE in dev_tm_dirac_kappa we need the true mu (not 2 kappa mu!)
 
 // Init x,p,r for k=0
 // Allocate some numbers for host <-> device interaction
 hipMalloc((void **) &dotprod, sizeof(REAL));
 hipMalloc((void **) &dotprod2, sizeof(REAL));
 hipMalloc((void **) &rk, sizeof(REAL));
 hipMalloc((void **) &alpha, sizeof(REAL));
 hipMalloc((void **) &beta, sizeof(REAL));
 printf("%s\n", hipGetErrorString(hipGetLastError())); 
 
 
 //init blas
 cublasInit();
 printf("%s\n", hipGetErrorString(hipGetLastError())); 
 printf("have initialized cublas\n");
 
 
 // go over to kappa (if wanted)
 REAL scaleparam = sqrt(1.0/(2.0 * (REAL)hostkappa));
 printf("1/2kappa = %.8f\n",scaleparam);
 //dev_skalarmult_spinor_field<<<griddim2, blockdim2 >>>(spinin,scaleparam, spin1);
 //dev_copy_spinor_field<<<griddim2, blockdim2 >>>(spin1, spinin);
 
 
 dev_copy_spinor_field<<<griddim2, blockdim2 >>>(spinin, spin0);
 dev_zero_spinor_field<<<griddim2, blockdim2 >>>(spin1); // x_0 = 0
 dev_copy_spinor_field<<<griddim2, blockdim2 >>>(spinin, spin2);
 dev_zero_spinor_field<<<griddim2, blockdim2 >>>(spin3);
 printf("%s\n", hipGetErrorString(hipGetLastError()));
 
 
 
 
 //relative precision -> get initial residue
 sourcesquarenorm = hipblasSdot (24*VOLUME, (const float *)spinin, 1, (const float *)spinin, 1);
 host_rk = sourcesquarenorm; //for use in main loop
 printf("Squarenorm Source:\t%.8e\n", sourcesquarenorm);
 printf("%s\n", hipGetErrorString(hipGetLastError()));
 
  printf("Entering cg-loop\n");
 for(i=0;i<maxit;i++){ //MAIN LOOP
  
  // D Ddagger    --   Ddagger = gamma5 D gamma5  for Wilson Dirac Operator
  // mu -> -mu for twisted term
  // DO NOT USE tm_dirac_dagger_kappa here, otherwise spin2 will be overwritten!!!
  
  unbind_texture_spin(1);
     // GAMMA5, mu -> -mu
     dev_gamma5 <<<griddim2, blockdim2 >>> (spin2,spin4);
     dev_swapmu <<<1,1>>> ();
  bind_texture_spin(spin4,1);
     //D_tm 
     dev_tm_dirac_kappa <<<griddim3, blockdim3 >>> (gf, spin4, spin3, dev_nn);
  unbind_texture_spin(1);
     //GAMMA5 mu -> -mu
     dev_gamma5 <<<griddim2, blockdim2 >>>(spin3,spin4);
     dev_swapmu <<<1,1>>> ();
  bind_texture_spin(spin4,1);
     //D_tm
     dev_tm_dirac_kappa <<<griddim3, blockdim3 >>> (gf, spin4, spin3, dev_nn);
  
  //Here we have used the output spinor (spinout) to temporarly take the field and to 
  //copy it to the texture field (spin4)!!

  
 //alpha
  host_dotprod = hipblasSdot (24*VOLUME, (const float *) spin2, 1,
            (const float *) spin3, 1);
  host_alpha = (host_rk / host_dotprod); // alpha = r*r/ p M p
   
 //r(k+1)
 hipblasSaxpy (24*VOLUME,-1.0*host_alpha, (const float *) spin3, 1, (float *) spin0, 1);  

 //x(k+1);
 hipblasSaxpy (24*VOLUME, host_alpha, (const float *) spin2,  1, (float *) spin1, 1);

 printf("%s\n", hipGetErrorString(hipGetLastError()));

  //Abbruch?
  host_dotprod = hipblasSdot (24*VOLUME, (const float *) spin0, 1,(const float *) spin0, 1);
  
 if ((host_dotprod <= eps*sourcesquarenorm)){//error-limit erreicht
   break; 
 }
  printf("iter %d: err = %.8e\n", i, host_dotprod);
  
 //beta
 host_beta =host_dotprod/host_rk;
 //p(k+1)
 hipblasSscal (24*VOLUME, host_beta, (float *)spin2, 1);
 hipblasSaxpy (24*VOLUME, 1.0, (const float *) spin0,  1, (float *) spin2, 1);

 host_rk = host_dotprod;
 
 // recalculate residue frome r = b - Ax
 if(((i+1) % N_recalcres) == 0){
    // r_(k+1) = Ax -b 
    printf("Recalculating residue\n");
    
    // D Ddagger   --   Ddagger = gamma5 D gamma5  for Wilson Dirac Operator
    // DO NOT USE tm_dirac_dagger_kappa here, otherwise spin2 will be overwritten!!!
      
      //GAMMA5
    unbind_texture_spin(1);
      dev_gamma5 <<<griddim2, blockdim2 >>> (spin1,spin4);
      dev_swapmu <<<1,1>>> ();
    bind_texture_spin(spin4,1);
   
      //D_tm GAMMA5, mu -> -mu
      dev_tm_dirac_kappa <<<griddim3, blockdim3 >>> (gf, spin4, spin3, dev_nn);
      dev_gamma5 <<<griddim2, blockdim2 >>>(spin3,spinout);
      dev_swapmu <<<1,1>>> ();
  
    //printf("Unbinding texture of spinorfield\n");
    unbind_texture_spin(1);
    hipMemcpy(spin4, spinout,size2, hipMemcpyDeviceToDevice);
    //printf("Rebinding texture to spinorfield\n");
    bind_texture_spin(spin4,1);
      
      //D_tm
      dev_tm_dirac_kappa<<<griddim3, blockdim3 >>>(gf, spin4, spin3, dev_nn);
    
    // r = b - Ax
    hipblasSscal (24*VOLUME, -1.0, (float *)spin3, 1);
    hipblasSaxpy (24*VOLUME, 1.0, (const float *) spinin,  1, (float *) spin3, 1);
    hipblasScopy (24*VOLUME, (const float *)spin3, 1, (float *)spin0, 1);
    
    //dev_skalarmult_add_assign_spinor_field<<<griddim2, blockdim2 >>>(spinin, -1.0, spin3, spin0);
   }//recalculate residue

 }//MAIN LOOP cg	
  
  
  printf("Final residue: %.6e\n",host_dotprod);
  // x_result = spin1 !
  
 if(rescalekappa == 1){  //want D^-1 rescaled by 2*kappa
  
//multiply with D^dagger
    unbind_texture_spin(1);
      dev_gamma5 <<<griddim2, blockdim2 >>> (spin1,spin4);
      dev_swapmu <<<1,1>>> ();
    bind_texture_spin(spin4,1);
      dev_tm_dirac_kappa <<<griddim3, blockdim3 >>> (gf, spin4, spin3, dev_nn);
      dev_gamma5 <<<griddim2, blockdim2 >>>(spin3,spin1);
      dev_swapmu <<<1,1>>> ();
    unbind_texture_spin(1);


 //go over to non-kappa, Ddagger = g5 D g5
 dev_skalarmult_spinor_field<<<griddim2, blockdim2 >>>(spin1,1.0/(scaleparam*scaleparam), spinout);  
 
  // times operator == source ?? 
  //dev_tm_dirac_kappa<<<griddim3, blockdim3 >>>(gf, spin3, spinout, nn_grid);
  }
  else{
   dev_copy_spinor_field<<<griddim2, blockdim2 >>>(spin1,spinout);
  }
  
  
  unbind_texture_gf();
  hipFree(dotprod);
  hipFree(dotprod2);
  hipFree(rk);
  hipFree(alpha);
  hipFree(beta);
  cublasShutdown();
}





// this is the eo version of the device cg inner solver 
// we invert the hermitean Q_{-} Q_{+}
extern "C" void dev_cg_eo(dev_su3_2v * gf,dev_spinor* spinin, dev_spinor* spinout, 
dev_spinor* spin0, dev_spinor* spin1, dev_spinor* spin2, dev_spinor* spin3, dev_spinor* spin4, int *grid, int * nn_grid, REAL* output,REAL* erg, int xsize, int ysize, int rescalekappa){
 
 
 REAL host_alpha, host_beta, host_dotprod, host_rk, sourcesquarenorm;
 REAL * dotprod, * dotprod2, * rk, * alpha, *beta;
 
 
 
 int i, gridsize;
 int maxit = max_innersolver_it;
 REAL eps = (REAL) innersolver_precision;
 int N_recalcres = 10; // after N_recalcres iterations calculate r = A x_k - b
 
 

 dim3 blockdim(1,1);
 dim3 blockdim2(128,1,1);
 if( VOLUME/2 >= 128){
   gridsize =VOLUME/2/128;
 }
 else{
   gridsize=1;
 }
 dim3 griddim2(gridsize,1,1);
 
 
 if((VOLUME/2)%BLOCK != 0){
   printf("Error: VOLUME/2 is not a multiple of BLOCK. Aborting...\n");
   exit(100);
 }
 int blockdim3=BLOCK;
 if( VOLUME/2 >= BLOCK){
   gridsize =VOLUME/2/BLOCK;
 }
 else{
   gridsize=1;
 }
 int griddim3=gridsize; 
 
 size_t size2 = sizeof(float4)*6*VOLUME/2;
 
 //Bind texture gf
 bind_texture_gf(gf);
 
 
 //Initialize some stuff
  printf("mu = %f\n", g_mu);
  dev_complex h0,h1,h2,h3;
  h0.re = (REAL)ka0.re;    h0.im = -(REAL)ka0.im;
  h1.re = (REAL)ka1.re;    h1.im = -(REAL)ka1.im;
  h2.re = (REAL)ka2.re;    h2.im = -(REAL)ka2.im;
  h3.re = (REAL)ka3.re;    h3.im = -(REAL)ka3.im;
  he_cg_init<<< 1, 1 >>> (grid, (REAL) g_kappa, (REAL)(g_mu/(2.0*g_kappa)), h0,h1,h2,h3);
  // BEWARE in dev_tm_dirac_kappa we need the true mu (not 2 kappa mu!)
 
 // Init x,p,r for k=0
 // Allocate some numbers for host <-> device interaction
 hipMalloc((void **) &dotprod, sizeof(REAL));
 hipMalloc((void **) &dotprod2, sizeof(REAL));
 hipMalloc((void **) &rk, sizeof(REAL));
 hipMalloc((void **) &alpha, sizeof(REAL));
 hipMalloc((void **) &beta, sizeof(REAL));
 printf("%s\n", hipGetErrorString(hipGetLastError())); 
 
 
 //init blas
 cublasInit();
 printf("%s\n", hipGetErrorString(hipGetLastError())); 
 printf("have initialized cublas\n");
 
 
 // go over to kappa (if wanted)
 REAL scaleparam = sqrt(1.0/(2.0 * (REAL)hostkappa));
 printf("1/2kappa = %.8f\n",scaleparam);
 //dev_skalarmult_spinor_field<<<griddim2, blockdim2 >>>(spinin,scaleparam, spin1);
 //dev_copy_spinor_field<<<griddim2, blockdim2 >>>(spin1, spinin);
 
 
 dev_copy_spinor_field<<<griddim2, blockdim2 >>>(spinin, spin0);
 dev_zero_spinor_field<<<griddim2, blockdim2 >>>(spin1); // x_0 = 0
 dev_copy_spinor_field<<<griddim2, blockdim2 >>>(spinin, spin2);
 dev_zero_spinor_field<<<griddim2, blockdim2 >>>(spin3);
 printf("%s\n", hipGetErrorString(hipGetLastError()));
 
 
 
 //relative precision -> get initial residue
 sourcesquarenorm = hipblasSdot (24*VOLUME/2, (const float *)spinin, 1, (const float *)spinin, 1);
 host_rk = sourcesquarenorm; //for use in main loop
 printf("Squarenorm Source:\t%.8e\n", sourcesquarenorm);
 printf("%s\n", hipGetErrorString(hipGetLastError()));
 
  printf("Entering cg-loop\n");
 for(i=0;i<maxit;i++){ //MAIN LOOP
  
  // Q_{-}Q{+}
  dev_Qtm_pm_psi(spin2, spin3, griddim3, blockdim3);
  
  
 //alpha
  host_dotprod = hipblasSdot (24*VOLUME/2, (const float *) spin2, 1,
            (const float *) spin3, 1);
  host_alpha = (host_rk / host_dotprod); // alpha = r*r/ p M p
   
 //r(k+1)
 hipblasSaxpy (24*VOLUME/2,-1.0*host_alpha, (const float *) spin3, 1, (float *) spin0, 1);  

 //x(k+1);
 hipblasSaxpy (24*VOLUME/2, host_alpha, (const float *) spin2,  1, (float *) spin1, 1);

 printf("%s\n", hipGetErrorString(hipGetLastError()));

  //Abbruch?
  host_dotprod = hipblasSdot (24*VOLUME/2, (const float *) spin0, 1,(const float *) spin0, 1);
  
 if ((host_dotprod <= eps*sourcesquarenorm)){//error-limit erreicht
   break; 
 }
  printf("iter %d: err = %.8e\n", i, host_dotprod);
  
 //beta
 host_beta =host_dotprod/host_rk;
 //p(k+1)
 hipblasSscal (24*VOLUME/2, host_beta, (float *)spin2, 1);
 hipblasSaxpy (24*VOLUME/2, 1.0, (const float *) spin0,  1, (float *) spin2, 1);

 host_rk = host_dotprod;
 
 // recalculate residue frome r = b - Ax
 if(((i+1) % N_recalcres) == 0){
    // r_(k+1) = Ax -b 
    printf("Recalculating residue\n");
    
    // D Ddagger   --   Ddagger = gamma5 D gamma5  for Wilson Dirac Operator
    // DO NOT USE tm_dirac_dagger_kappa here, otherwise spin2 will be overwritten!!!
      
    // Q_{-}Q{+}
    dev_Qtm_pm_psi(spin1, spin3, griddim3, blockdim3);
      
        
    
    // r = b - Ax
    hipblasSscal (24*VOLUME/2, -1.0, (float *)spin3, 1);
    hipblasSaxpy (24*VOLUME/2, 1.0, (const float *) spinin,  1, (float *) spin3, 1);
    hipblasScopy (24*VOLUME/2, (const float *)spin3, 1, (float *)spin0, 1);
    //dev_skalarmult_add_assign_spinor_field<<<griddim2, blockdim2 >>>(spinin, -1.0, spin3, spin0);
   }//recalculate residue

 }//MAIN LOOP cg	
  
  
  printf("Final residue: %.6e\n",host_dotprod);
  // x_result = spin1 !
  
  //no multiplication with D^{dagger} here and no return to non-kappa basis as in dev_cg!
  dev_copy_spinor_field<<<griddim2, blockdim2 >>>(spin1,spinout);
  
  
  unbind_texture_gf();
  hipFree(dotprod);
  hipFree(dotprod2);
  hipFree(rk);
  hipFree(alpha);
  hipFree(beta);
  cublasShutdown();
}












//initialize nearest-neighbour table for gpu
void initnn(){
  int t,x,y,z,pos;
  for(t=0;t<T;t++){
   for(x=0; x<LX; x++){
    for(y=0; y<LY; y++){
     for(z=0; z<LZ; z++){   
          pos= z + LZ*(y + LY*(x + LX*t));
          //plus direction
          nn[8*pos+0] = z + LZ*(y + LY*(x + LX*((t+1)%T)));
          nn[8*pos+1] = z + LZ*(y + LY*((x+1)%LX + LX*t));
          nn[8*pos+2] = z + LZ*((y+1)%LY + LY*(x + LX*t));
          nn[8*pos+3] = (z+1)%LZ + LX*(y + LY*(x + LX*t));
          //minus direction
          if(t==0){
            nn[8*pos+4] = z + LZ*(y + LY*(x + LX*((T-1))));
          }
          else{
            nn[8*pos+4] = z + LZ*(y + LY*(x + LX*((t-1))));
          }
          if(x==0){
            nn[8*pos+5] = z + LZ*(y + LY*((LX-1) + LX*t));
          }
          else{
            nn[8*pos+5] = z + LZ*(y + LY*((x-1) + LX*t));
          }
          if(y==0){
            nn[8*pos+6] = z + LZ*((LY-1) + LY*(x + LX*t));
          }
          else{
            nn[8*pos+6] = z + LZ*((y-1) + LY*(x + LX*t));
          }
          if(z==0){
            nn[8*pos+7] = (LZ-1) + LZ*(y + LY*(x + LX*t));
          }
          else{
            nn[8*pos+7] = (z-1) + LZ*(y + LY*(x + LX*t));
          }          
        }
      }
    } 
  }
}





//initialize nearest-neighbour table for gpu with even-odd enabled
//init_nn must have been called before for initialization of nn
void initnn_eo(){
  int x,y,z,t,ind,nnpos,j;
  int evenpos=0;
  int oddpos=0;
  for(t=0;t<T;t++){
    for(x=0;x<LX;x++){
      for(y=0;y<LY;y++){
        for(z=0;z<LZ;z++){
          ind = g_ipt[t][x][y][z];
          
          if(((t+x+y+z)%2 == 0)){
            nnpos = g_lexic2eosub[ind];
            for(j=0;j<4;j++){
              nn_eo[8*nnpos+j] = g_lexic2eosub[ g_iup[ind][j] ];
            }
            for(j=0;j<4;j++){
              nn_eo[8*nnpos+4+j] = g_lexic2eosub[ g_idn[ind][j] ];
            }
            eoidx_even[evenpos] = ind;
            evenpos++;
          }
          else{
            nnpos = g_lexic2eosub[ind];
            for(j=0;j<4;j++){
              nn_oe[8*nnpos+j] = g_lexic2eosub[ g_iup[ind][j] ];
            }
            for(j=0;j<4;j++){
              nn_oe[8*nnpos+4+j] = g_lexic2eosub[ g_idn[ind][j] ];
            }
            eoidx_odd[oddpos] = ind;
            oddpos++;
          }
        }
      }
    }
  }
}




// show the nn table eo
void shownn_eo(){
  int i,pos;
  printf("eo part\n");
  for(pos=0;pos<VOLUME/2;pos++){ 
       printf("p=%d\t", pos);
       for(i=0;i<8;i++){
          printf("%d ",nn_eo[8*pos+i]);
          //lptovec(nn[8*pos+i]);
        }
        printf("\n");
    }
  printf("oe part\n");
  for(pos=0;pos<VOLUME/2;pos++){ 
       printf("p=%d\t", pos);
       for(i=0;i<8;i++){
          printf("%d ",nn_oe[8*pos+i]);
          //lptovec(nn[8*pos+i]);
        }
        printf("\n");
    }
    
  printf("site index even\n");
  for(pos=0;pos<VOLUME/2;pos++){ 
       printf("p=%d\t", pos);
          printf("%d ",eoidx_even[pos]);
          //lptovec(nn[8*pos+i]);
        printf("\n");
  }

  printf("site index odd\n");
  for(pos=0;pos<VOLUME/2;pos++){ 
       printf("p=%d\t", pos);
          printf("%d ",eoidx_odd[pos]);
          //lptovec(nn[8*pos+i]);
        printf("\n");
  }
  printf("checking forward even\n");
  for(pos=0;pos<VOLUME/2;pos++){
    for(i=0;i<4;i++){
      printf("%d = %d\n",pos, nn_oe[8*nn_eo[8*pos+i]+4+i]);
    }
  }

  printf("checking backward even\n");
  for(pos=0;pos<VOLUME/2;pos++){
    for(i=0;i<4;i++){
      printf("%d = %d\n",pos, nn_oe[8*nn_eo[8*pos+4+i]+i]);
    }
  }

  printf("checking forward odd\n");
  for(pos=0;pos<VOLUME/2;pos++){
    for(i=0;i<4;i++){
      printf("%d = %d\n",pos, nn_eo[8*nn_oe[8*pos+i]+4+i]);
    }
  }

  printf("checking backward odd\n");
  for(pos=0;pos<VOLUME/2;pos++){
    for(i=0;i<4;i++){
      printf("%d = %d\n",pos, nn_eo[8*nn_oe[8*pos+4+i]+i]);
    }
  }
}





void lptovec(int k){
  int L3 = L*L*L;
  int L2 = L*L;
  int x0,x1,x2,x3;
  x0 = k/L3;
  k = k-x0*L3; 
  x3 = k/L2;
  k = k-x3*L2;
  x2 = k/L;
  k = k-x2*L;
  x1 = k;
  printf("%d,%d,%d,%d;  ",x0,x3,x2,x1);
}


// show nn table 
void shownn(){
  int t,x,y,z,i,pos;
  int lx,ly,lz,lt;
    lx = LX;
    ly = LY;
    lz = LZ;
    lt =T;  
  for(t=0;t<lt;t++){ 
    for(x=0; x<lx; x++){
      for(y=0; y<ly; y++){
        for(z=0; z<lz; z++){
          pos= z + lz*(y + ly*(x + lx*t));
          printf("p=%d\t", pos);
          for(i=0;i<8;i++){
            printf("%d ",nn[8*pos+i]);
            //lptovec(nn[8*pos+i]);
          }
          printf("\n");
          //compare with geometry fields of hmc
          //might NOT WORK for even-odd? What are geometry indices in case of even-odd?
          printf("%d: %d %d %d %d %d %d %d %d\n",g_ipt[t][x][y][z],g_iup[pos][0],g_iup[pos][1],g_iup[pos][2],g_iup[pos][3],g_idn[pos][0],g_idn[pos][1],g_idn[pos][2],g_idn[pos][3]);
        }
      }
    }
  }
}




// get 2 first rows of gf float4 type
//  
//
void su3to2vf4(su3** gf, dev_su3_2v* h2d_gf){
  int i,j;
  for (i=0;i<VOLUME;i++){
   for(j=0;j<4;j++){
   //first row
    h2d_gf[3*(4*i+j)].x = (REAL) gf[i][j].c00.re;
    h2d_gf[3*(4*i+j)].y = (REAL) gf[i][j].c00.im;
    h2d_gf[3*(4*i+j)].z = (REAL) gf[i][j].c01.re;
    h2d_gf[3*(4*i+j)].w = (REAL) gf[i][j].c01.im;
    h2d_gf[3*(4*i+j)+1].x = (REAL) gf[i][j].c02.re;
    h2d_gf[3*(4*i+j)+1].y = (REAL) gf[i][j].c02.im;      
   //second row
    h2d_gf[3*(4*i+j)+1].z = (REAL) gf[i][j].c10.re;
    h2d_gf[3*(4*i+j)+1].w = (REAL) gf[i][j].c10.im;
    h2d_gf[3*(4*i+j)+2].x = (REAL) gf[i][j].c11.re;
    h2d_gf[3*(4*i+j)+2].y = (REAL) gf[i][j].c11.im;
    h2d_gf[3*(4*i+j)+2].z = (REAL) gf[i][j].c12.re;
    h2d_gf[3*(4*i+j)+2].w = (REAL) gf[i][j].c12.im;      
  } 
 }
}







// this is to reconstruc the gf on the host from 2 rows of the link
// may be used for tests
void reconstructgf_2v (dev_su3* gf){
  complex help1;
  complex help2;
  //third row from cconj(cross product of first and second row)
  _mult_assign_complex(help1,(*gf)[0][1],(*gf)[1][2]);
  _mult_assign_complex(help2,(*gf)[0][2],(*gf)[1][1]);
  _diff_complex(help1,help2);
  help1.im = -help1.im;
  (*gf)[2][0].re = help1.re;
  (*gf)[2][0].im = help1.im;
  
  _mult_assign_complex(help1,(*gf)[0][2],(*gf)[1][0]);
  _mult_assign_complex(help2,(*gf)[0][0],(*gf)[1][2]);
  _diff_complex(help1,help2);
  help1.im = -help1.im;
  (*gf)[2][1].re = help1.re;
  (*gf)[2][1].im = help1.im;
  
  _mult_assign_complex(help1,(*gf)[0][0],(*gf)[1][1]);
  _mult_assign_complex(help2,(*gf)[0][1],(*gf)[1][0]);
  _diff_complex(help1,help2);
  help1.im = -help1.im;
  (*gf)[2][2].re = help1.re;
  (*gf)[2][2].im = help1.im;
  return;
}



void showcompare_gf(int t, int x, int y, int z, int mu){
   int ind1 = g_ipt[t][x][y][z];
   su3 ** gf1 = g_gauge_field;
   printf("(%f,%f)\t(%f,%f)\t(%f,%f)\n",gf1[ind1][mu].c00.re,
   					gf1[ind1][mu].c00.im,
   					gf1[ind1][mu].c01.re,
   					gf1[ind1][mu].c01.im,
   					gf1[ind1][mu].c02.re,
   					gf1[ind1][mu].c02.im
   );
   printf("(%f,%f)\t(%f,%f)\t(%f,%f)\n",gf1[ind1][mu].c10.re,
   					gf1[ind1][mu].c10.im,
   					gf1[ind1][mu].c11.re,
   					gf1[ind1][mu].c11.im,
   					gf1[ind1][mu].c12.re,
   					gf1[ind1][mu].c12.im
   );
   printf("(%f,%f)\t(%f,%f)\t(%f,%f)\n",gf1[ind1][mu].c20.re,
   					gf1[ind1][mu].c20.im,
   					gf1[ind1][mu].c21.re,
   					gf1[ind1][mu].c21.im,
   					gf1[ind1][mu].c22.re,
   					gf1[ind1][mu].c22.im
   );
   printf("\n\n");

   int ind2 =  z + LZ*(y + LY*(x + LX*t));
   printf("(%f,%f)\t(%f,%f)\t(%f,%f)\n",h2d_gf[3*(4*ind2+mu)].x,
   					h2d_gf[3*(4*ind2+mu)].y,
   					h2d_gf[3*(4*ind2+mu)].z,
   					h2d_gf[3*(4*ind2+mu)].w,
   					h2d_gf[3*(4*ind2+mu)+1].x,
   					h2d_gf[3*(4*ind2+mu)+1].y
   );
   printf("(%f,%f)\t(%f,%f)\t(%f,%f)\n",h2d_gf[3*(4*ind2+mu)+1].z,
   					h2d_gf[3*(4*ind2+mu)+1].w,
   					h2d_gf[3*(4*ind2+mu)+2].x,
   					h2d_gf[3*(4*ind2+mu)+2].y,
   					h2d_gf[3*(4*ind2+mu)+2].z,
   					h2d_gf[3*(4*ind2+mu)+2].w
   );   
   
   dev_su3 help;
   
   help[0][0].re = h2d_gf[3*(4*ind2+mu)].x;
   help[0][0].im = h2d_gf[3*(4*ind2+mu)].y;
   help[0][1].re = h2d_gf[3*(4*ind2+mu)].z;
   help[0][1].im = h2d_gf[3*(4*ind2+mu)].w;

   help[0][2].re = h2d_gf[3*(4*ind2+mu)+1].x;
   help[0][2].im = h2d_gf[3*(4*ind2+mu)+1].y;
   help[1][0].re = h2d_gf[3*(4*ind2+mu)+1].z;
   help[1][0].im = h2d_gf[3*(4*ind2+mu)+1].w;
   
   help[1][1].re = h2d_gf[3*(4*ind2+mu)+2].x;
   help[1][1].im = h2d_gf[3*(4*ind2+mu)+2].y;
   help[1][2].re = h2d_gf[3*(4*ind2+mu)+2].z;
   help[1][2].im = h2d_gf[3*(4*ind2+mu)+2].w;   
   
   reconstructgf_2v (&help); 

   printf("(%f,%f)\t(%f,%f)\t(%f,%f)\n",help[2][0].re,
   					help[2][0].im,
   					help[2][1].re,
   					help[2][1].im,
   					help[2][2].re,
   					help[2][2].im
   );
     
}







// convert spinor to double 
void convert2double_spin(dev_spinor* spin, spinor* h2d){
  int i,Vol;
  if(even_odd_flag){
    Vol = VOLUME/2;
  }
  else{
    Vol = VOLUME;
  }
  for (i=0;i<Vol;i++){
  
        h2d[i].s0.c0.re = (double) spin[6*i+0].x;
        h2d[i].s0.c0.im = (double) spin[6*i+0].y;
        h2d[i].s0.c1.re = (double) spin[6*i+0].z;
        h2d[i].s0.c1.im = (double) spin[6*i+0].w;
        
        h2d[i].s0.c2.re = (double) spin[6*i+1].x;
        h2d[i].s0.c2.im = (double) spin[6*i+1].y;
        h2d[i].s1.c0.re = (double) spin[6*i+1].z;
        h2d[i].s1.c0.im = (double) spin[6*i+1].w;   
        
        h2d[i].s1.c1.re = (double) spin[6*i+2].x;
        h2d[i].s1.c1.im = (double) spin[6*i+2].y;
        h2d[i].s1.c2.re = (double) spin[6*i+2].z;
        h2d[i].s1.c2.im = (double) spin[6*i+2].w;  
        
        h2d[i].s2.c0.re = (double) spin[6*i+3].x;
        h2d[i].s2.c0.im = (double) spin[6*i+3].y;
        h2d[i].s2.c1.re = (double) spin[6*i+3].z;
        h2d[i].s2.c1.im = (double) spin[6*i+3].w;  
        
        h2d[i].s2.c2.re = (double) spin[6*i+4].x;
        h2d[i].s2.c2.im = (double) spin[6*i+4].y;
        h2d[i].s3.c0.re = (double) spin[6*i+4].z;
        h2d[i].s3.c0.im = (double) spin[6*i+4].w; 
        
        h2d[i].s3.c1.re = (double) spin[6*i+5].x;
        h2d[i].s3.c1.im = (double) spin[6*i+5].y;
        h2d[i].s3.c2.re = (double) spin[6*i+5].z;
        h2d[i].s3.c2.im = (double) spin[6*i+5].w; 
        
  }
}





// convert spinor to REAL4 (float4, double4) 
void convert2REAL4_spin(spinor* spin, dev_spinor* h2d){
  int i,Vol;
  if(even_odd_flag){
    Vol = VOLUME/2;
  }
  else{
    Vol = VOLUME;
  }
  for (i=0;i<Vol;i++){
    
        h2d[6*i+0].x = (REAL) spin[i].s0.c0.re;
        h2d[6*i+0].y = (REAL) spin[i].s0.c0.im;
        h2d[6*i+0].z = (REAL) spin[i].s0.c1.re;
        h2d[6*i+0].w = (REAL) spin[i].s0.c1.im;
        
        h2d[6*i+1].x = (REAL) spin[i].s0.c2.re;
        h2d[6*i+1].y = (REAL) spin[i].s0.c2.im;
        h2d[6*i+1].z = (REAL) spin[i].s1.c0.re;
        h2d[6*i+1].w = (REAL) spin[i].s1.c0.im;
        
        h2d[6*i+2].x = (REAL) spin[i].s1.c1.re;
        h2d[6*i+2].y = (REAL) spin[i].s1.c1.im;
        h2d[6*i+2].z = (REAL) spin[i].s1.c2.re;
        h2d[6*i+2].w = (REAL) spin[i].s1.c2.im;
        
        h2d[6*i+3].x = (REAL) spin[i].s2.c0.re;
        h2d[6*i+3].y = (REAL) spin[i].s2.c0.im;
        h2d[6*i+3].z = (REAL) spin[i].s2.c1.re;
        h2d[6*i+3].w = (REAL) spin[i].s2.c1.im;
        
        h2d[6*i+4].x = (REAL) spin[i].s2.c2.re;
        h2d[6*i+4].y = (REAL) spin[i].s2.c2.im;
        h2d[6*i+4].z = (REAL) spin[i].s3.c0.re;
        h2d[6*i+4].w = (REAL) spin[i].s3.c0.im;
        
        h2d[6*i+5].x = (REAL) spin[i].s3.c1.re;
        h2d[6*i+5].y = (REAL) spin[i].s3.c1.im;
        h2d[6*i+5].z = (REAL) spin[i].s3.c2.re;
        h2d[6*i+5].w = (REAL) spin[i].s3.c2.im;
    
  }
}





void init_mixedsolve(su3** gf){
hipError_t cudaerr;

  
  /* allocate 2 rows of gf = 3*4*VOLUME float4's*/
  size_t dev_gfsize = 3*4*VOLUME * sizeof(dev_su3_2v); 
  
  if((cudaerr=hipMalloc((void **) &dev_gf, dev_gfsize)) != hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of gauge field failed. Aborting...\n");
    exit(200);
  }   // Allocate array on device
  else{
    printf("Allocated gauge field on device\n");
  }  
  
  
  h2d_gf = (dev_su3_2v *)malloc(dev_gfsize); // Allocate REAL conversion gf on host
  su3to2vf4(gf,h2d_gf);
  hipMemcpy(dev_gf, h2d_gf, dev_gfsize, hipMemcpyHostToDevice);


//grid 
  size_t nnsize = 8*VOLUME*sizeof(int);
  nn = (int *) malloc(nnsize);
  hipMalloc((void **) &dev_nn, nnsize);
  
  initnn();
  //shownn();
  //showcompare_gf(T-1, LX-1, LY-1, LZ-1, 3);
  hipMemcpy(dev_nn, nn, nnsize, hipMemcpyHostToDevice);
  
  //free again
  free(nn);


// Spinors
  size_t dev_spinsize = 6*VOLUME * sizeof(dev_spinor); /* float4 */

  if((void*)(h2d_spin = (dev_spinor *)malloc(dev_spinsize)) == NULL){
    printf("Could not allocate memory for h2d_spin. Aborting...\n");
    exit(200);
  } // Allocate float conversion spinor on host
  
  hipMalloc((void **) &dev_spin1, dev_spinsize);   // Allocate array spin1 on device
  hipMalloc((void **) &dev_spin2, dev_spinsize);   // Allocate array spin2 on device
  hipMalloc((void **) &dev_spin3, dev_spinsize);   // Allocate array spin3 on device
  hipMalloc((void **) &dev_spin4, dev_spinsize);
  hipMalloc((void **) &dev_spin5, dev_spinsize);
  hipMalloc((void **) &dev_spinin, dev_spinsize);
  hipMalloc((void **) &dev_spinout, dev_spinsize);
  if((cudaerr=hipGetLastError())!=hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of spinor fields failed. Aborting...\n");
    exit(200);
  }
  else{
    printf("Allocated spinor fields on device\n");
  }
  
  
  output_size = LZ*T*sizeof(float); // parallel in t and z direction
  hipMalloc((void **) &dev_output, output_size);   // output array
  float * host_output = (float*) malloc(output_size);

  int grid[5];
  grid[0]=LX; grid[1]=LY; grid[2]=LZ; grid[3]=T; grid[4]=VOLUME;
 
  hipMalloc((void **) &dev_grid, 5*sizeof(int));
  hipMemcpy(dev_grid, &(grid[0]), 5*sizeof(int), hipMemcpyHostToDevice);
  
}






void init_mixedsolve_eo(su3** gf){
hipError_t cudaerr;
  dev_complex help;

  /* allocate 2 rows of gf = 3*4*VOLUME float4's*/
  size_t dev_gfsize = 3*4*VOLUME * sizeof(dev_su3_2v); 
  
  
  
  if((cudaerr=hipMalloc((void **) &dev_gf, dev_gfsize)) != hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of gauge field failed. Aborting...\n");
    exit(200);
  }   // Allocate array on device
  else{
    printf("Allocated gauge field on device\n");
  }  
  
  
  h2d_gf = (dev_su3_2v *)malloc(dev_gfsize); // Allocate REAL conversion gf on host
  su3to2vf4(gf,h2d_gf);
  hipMemcpy(dev_gf, h2d_gf, dev_gfsize, hipMemcpyHostToDevice);



//grid 
  size_t nnsize = 8*VOLUME*sizeof(int);
  nn = (int *) malloc(nnsize);
  
  //nn grid for even-odd
  nn_eo = (int *) malloc(nnsize/2);
  nn_oe = (int *) malloc(nnsize/2);
  
  hipMalloc((void **) &dev_nn, nnsize);
  hipMalloc((void **) &dev_nn_eo, nnsize/2);
  hipMalloc((void **) &dev_nn_oe, nnsize/2);
  
  
  size_t idxsize = VOLUME/2*sizeof(int);
  eoidx_even = (int *) malloc(idxsize);
  eoidx_odd = (int *) malloc(idxsize);
  hipMalloc((void **) &dev_eoidx_even, idxsize);
  hipMalloc((void **) &dev_eoidx_odd, idxsize);
  
  initnn();
  initnn_eo();
  //shownn_eo();
  
  //shownn();
  //showcompare_gf(T-1, LX-1, LY-1, LZ-1, 3);
  hipMemcpy(dev_nn, nn, nnsize, hipMemcpyHostToDevice);
  hipMemcpy(dev_nn_eo, nn_eo, nnsize/2, hipMemcpyHostToDevice);
  hipMemcpy(dev_nn_oe, nn_oe, nnsize/2, hipMemcpyHostToDevice);
  hipMemcpy(dev_eoidx_even, eoidx_even, idxsize, hipMemcpyHostToDevice);
  hipMemcpy(dev_eoidx_odd, eoidx_odd, idxsize, hipMemcpyHostToDevice);
  
  //free again
  free(eoidx_odd);
  free(eoidx_even);
  free(nn_oe);
  free(nn_eo);
  free(nn);
  
// Spinors
  size_t dev_spinsize = 6*VOLUME/2 * sizeof(dev_spinor); /* float4 */

  if((void*)(h2d_spin = (dev_spinor *)malloc(dev_spinsize)) == NULL){
    printf("Could not allocate memory for h2d_spin. Aborting...\n");
    exit(200);
  } // Allocate float conversion spinor on host
  
  hipMalloc((void **) &dev_spin1, dev_spinsize);   // Allocate array spin1 on device
  hipMalloc((void **) &dev_spin2, dev_spinsize);   // Allocate array spin2 on device
  hipMalloc((void **) &dev_spin3, dev_spinsize);   // Allocate array spin3 on device
  hipMalloc((void **) &dev_spin4, dev_spinsize);
  hipMalloc((void **) &dev_spin5, dev_spinsize);
  hipMalloc((void **) &dev_spinin, dev_spinsize);
  hipMalloc((void **) &dev_spinout, dev_spinsize);
  
  hipMalloc((void **) &dev_spin_eo1, dev_spinsize);
  hipMalloc((void **) &dev_spin_eo2, dev_spinsize);
  
  if((cudaerr=hipGetLastError())!=hipSuccess){
    printf("Error in init_mixedsolve(): Memory allocation of spinor fields failed. Aborting...\n");
    exit(200);
  }
  else{
    printf("Allocated spinor fields on device\n");
  }
  
  
  output_size = LZ*T*sizeof(float); // parallel in t and z direction
  hipMalloc((void **) &dev_output, output_size);   // output array
  float * host_output = (float*) malloc(output_size);

  int grid[5];
  grid[0]=LX; grid[1]=LY; grid[2]=LZ; grid[3]=T; grid[4]=VOLUME/2; 
  // dev_VOLUME is half of VOLUME for eo
 
  hipMalloc((void **) &dev_grid, 5*sizeof(int));
  hipMemcpy(dev_grid, &(grid[0]), 5*sizeof(int), hipMemcpyHostToDevice);
  
  

}



void finalize_mixedsolve(){

  hipFree(dev_spin1);
  hipFree(dev_spin2);
  hipFree(dev_spin3);
  hipFree(dev_spin4);
  hipFree(dev_spin5);
  hipFree(dev_spinin);
  hipFree(dev_spinout);
  hipFree(dev_gf);
  hipFree(dev_grid);
  hipFree(dev_output);
  hipFree(dev_nn);
  
  if(even_odd_flag){
    hipFree(dev_spin_eo1);
    hipFree(dev_spin_eo2);
    hipFree(dev_eoidx_even);
    hipFree(dev_eoidx_odd);
    hipFree(dev_nn_eo);
    hipFree(dev_nn_oe);
  
  }
  
  
  
  free(h2d_spin);
  free(h2d_gf);
}







extern "C" int mixed_solve (spinor * const P, spinor * const Q, const int max_iter, 
	   double eps, const int rel_prec,const int N){
  
  // source in Q, initial solution in P (not yet implemented)
  double rk;
  int outercount=0;
  clock_t start, stop, startinner, stopinner; 
  double timeelapsed = 0.0;
  double sourcesquarenorm;
  int iter;
  
  size_t dev_spinsize = 6*VOLUME * sizeof(dev_spinor); // float4 
  init_mixedsolve(g_gauge_field);
  
  // Start timer
  assert((start = clock())!=-1);
  
  rk = square_norm(Q, N, 1);
  sourcesquarenorm = rk; // for relative precision
  assign(g_spinor_field[DUM_SOLVER],Q,N);
  printf("Initial residue: %.16e\n",rk);
  zero_spinor_field(g_spinor_field[DUM_SOLVER+1],  N);//spin2 = x_k
  zero_spinor_field(g_spinor_field[DUM_SOLVER+2],  N);
  printf("The VOLUME is: %d\n",N);
  
for(iter=0; iter<max_iter; iter++){

   printf("Applying double precision Dirac-Op...\n");
   
   Q_pm_psi_gpu(g_spinor_field[DUM_SOLVER+3], g_spinor_field[DUM_SOLVER+2]);
   diff(g_spinor_field[DUM_SOLVER],g_spinor_field[DUM_SOLVER],g_spinor_field[DUM_SOLVER+3],N);
    // r_k = b - D x_k
   
   rk = square_norm(g_spinor_field[DUM_SOLVER], N, 0);
   
   printf("Residue after %d inner solver iterations: %.18e\n",outercount,rk);
   if(((rk <= eps) && (rel_prec == 0)) || ((rk <= eps*sourcesquarenorm) && (rel_prec == 1)))
   {
     printf("Reached solver precision of eps=%.2e\n",eps);
     //multiply with D^dagger
     Q_minus_psi_gpu(g_spinor_field[DUM_SOLVER+3], g_spinor_field[DUM_SOLVER+1]);
     assign(P, g_spinor_field[DUM_SOLVER+3], N);
  

    stop = clock();
    timeelapsed = (double) (stop-start)/CLOCKS_PER_SEC;
    printf("Inversion done in mixed precision.\n Number of iterations in outer solver: %d\n Squared residue: %.8e\n Time elapsed: %.6e sec\n", outercount, rk, timeelapsed);
    finalize_mixedsolve();
    return(iter*max_innersolver_it);  // MAYBE ONE SHOULD KEEP TRACK OF REAL INNER SOLVER STEPS
   }
   

  //initialize spin fields on device
  convert2REAL4_spin(g_spinor_field[DUM_SOLVER],h2d_spin);
  
  hipMemcpy(dev_spinin, h2d_spin, dev_spinsize, hipMemcpyHostToDevice);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

   // solve in single prec on device
   // D p_k = r_k
   printf("Entering inner solver\n");
   assert((startinner = clock())!=-1);
   dev_cg(dev_gf, dev_spinin, dev_spinout, dev_spin1, dev_spin2, dev_spin3, dev_spin4, dev_spin5, dev_grid,dev_nn, dev_output,NULL, T, LZ,0);
   stopinner = clock();
   timeelapsed = (double) (stopinner-startinner)/CLOCKS_PER_SEC;
   printf("Inner solver done\nTime elapsed: %.6e sec\n", timeelapsed);
   
  
   // copy back
   hipMemcpy(h2d_spin, dev_spinout, dev_spinsize, hipMemcpyDeviceToHost);
   printf("%s\n", hipGetErrorString(hipGetLastError()));
   
   convert2double_spin(h2d_spin, g_spinor_field[DUM_SOLVER+2]);
   
   add(g_spinor_field[DUM_SOLVER+1],g_spinor_field[DUM_SOLVER+1],g_spinor_field[DUM_SOLVER+2],N);
   // x_(k+1) = x_k + p_k
   
   outercount ++;
    
}// outer loop 

     printf("Did NOT reach solver precision of eps=%.2e\n",eps);
     //multiply with D^dagger
     Q_minus_psi_gpu(g_spinor_field[DUM_SOLVER+3], g_spinor_field[DUM_SOLVER+1]);
     assign(P, g_spinor_field[DUM_SOLVER+3], N);
  

    stop = clock();
    timeelapsed = (double) (stop-start)/CLOCKS_PER_SEC;
    printf("Inversion done in mixed precision.\n Number of iterations in outer solver: %d\n Squared residue: %.8e\n Time elapsed: %.6e sec\n", outercount, rk, timeelapsed);

  return(-1);
}








extern "C" int mixed_solve_eo (spinor * const P, spinor * const Q, const int max_iter, 
	   double eps, const int rel_prec, const int N){
  
  // source in Q, initial solution in P (not yet implemented)
  double rk;
  int outercount=0;
  clock_t start, stop, startinner, stopinner; 
  double timeelapsed = 0.0;
  double sourcesquarenorm;
  int iter;
  
  
  size_t dev_spinsize = 6*VOLUME/2 * sizeof(dev_spinor); // float4 even-odd !
  init_mixedsolve_eo(g_gauge_field);
  
  // Start timer
  assert((start = clock())!=-1);
  
  rk = square_norm(Q, N, 1);
  sourcesquarenorm=rk; // for relative prec
  assign(g_spinor_field[DUM_SOLVER],Q,N);
  printf("Initial residue: %.16e\n",rk);
  zero_spinor_field(g_spinor_field[DUM_SOLVER+1],  N);//spin2 = x_k
  zero_spinor_field(g_spinor_field[DUM_SOLVER+2],  N);
  printf("The VOLUME/2 is: %d\n",N);
  
for(iter=0; iter<max_iter; iter++){

   printf("Applying double precision EO Dirac-Op Q_{-}Q{+}...\n");
   
   Qtm_pm_psi(g_spinor_field[DUM_SOLVER+3], g_spinor_field[DUM_SOLVER+2]);
   diff(g_spinor_field[DUM_SOLVER],g_spinor_field[DUM_SOLVER],g_spinor_field[DUM_SOLVER+3],N);
    // r_k = b - D x_k
   
   rk = square_norm(g_spinor_field[DUM_SOLVER], N, 0);
   
   printf("Residue after %d inner solver iterations: %.18e\n",outercount,rk);
   
   if(((rk <= eps) && (rel_prec == 0)) || ((rk <= eps*sourcesquarenorm) && (rel_prec == 1)))
   {
     printf("Reached solver precision of eps=%.2e\n",eps);
     //multiply with D^dagger
     Qtm_minus_psi(g_spinor_field[DUM_SOLVER+3], g_spinor_field[DUM_SOLVER+1]);
     assign(P, g_spinor_field[DUM_SOLVER+3], N);
  

     stop = clock();
     timeelapsed = (double) (stop-start)/CLOCKS_PER_SEC;
     printf("EO Inversion done in mixed precision.\n Number of iterations in outer solver: %d\n Squared residue: %.8e\n Time elapsed: %.6e sec\n", outercount, rk, timeelapsed);
   
     finalize_mixedsolve();
     return(iter*max_innersolver_it);  // MAYBE ONE SHOULD KEEP TRACK OF REAL INNER SOLVER STEPS
   }
   
  //initialize spin fields on device
  convert2REAL4_spin(g_spinor_field[DUM_SOLVER],h2d_spin);
  
  hipMemcpy(dev_spinin, h2d_spin, dev_spinsize, hipMemcpyHostToDevice);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

   // solve in single prec on device
   // D p_k = r_k
   printf("Entering inner solver\n");
   assert((startinner = clock())!=-1);
   dev_cg_eo(dev_gf, dev_spinin, dev_spinout, dev_spin1, dev_spin2, dev_spin3, dev_spin4, dev_spin5, dev_grid,dev_nn, dev_output,NULL, T, LZ,0);
   stopinner = clock();
   timeelapsed = (double) (stopinner-startinner)/CLOCKS_PER_SEC;
   printf("Inner solver done\nTime elapsed: %.6e sec\n", timeelapsed);
   
  
   // copy back
   hipMemcpy(h2d_spin, dev_spinout, dev_spinsize, hipMemcpyDeviceToHost);
   printf("%s\n", hipGetErrorString(hipGetLastError()));
   
   convert2double_spin(h2d_spin, g_spinor_field[DUM_SOLVER+2]);
   
   add(g_spinor_field[DUM_SOLVER+1],g_spinor_field[DUM_SOLVER+1],g_spinor_field[DUM_SOLVER+2],N);
   // x_(k+1) = x_k + p_k
   
   outercount ++;
    
}// outer loop 
    
     printf("Did NOT reach solver precision of eps=%.2e\n",eps);
     //multiply with D^dagger
     Q_minus_psi_gpu(g_spinor_field[DUM_SOLVER+3], g_spinor_field[DUM_SOLVER+1]);
     assign(P, g_spinor_field[DUM_SOLVER+3], N);
  

    stop = clock();
    timeelapsed = (double) (stop-start)/CLOCKS_PER_SEC;
    printf("Inversion done in mixed precision.\n Number of iterations in outer solver: %d\n Squared residue: %.8e\n Time elapsed: %.6e sec\n", outercount, rk, timeelapsed);


  finalize_mixedsolve();
  return(-1);
}







